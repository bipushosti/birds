#include "hip/hip_runtime.h"

//Needs Header Files for the functions; The header file should have both C and CUDA functions



//This file uses 6 hourly data. Each day is 6 hours long and skipping a day means to add 6
//to the counter that counts the timesteps (l).

//The birds start at 00:00 UTC which is 6pm in central time examplewhen there is no day light savings
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>

#include <pthread.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <math.h>
#include <float.h>

#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <getopt.h>

#include <math.h>


//#include "birds_CUDA.h"
//#define CUDA_API_PER_THREAD_DEFAULT_STREAM

/*
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
*/

#define PI 			3.14159
#define LONG_SIZE		429
#define LAT_SIZE		429
#define LINESIZE		15*LONG_SIZE+LONG_SIZE - 3
#define TOTAL_DAYS		122
#define TIMESTEPS_PER_DAY	24
#define TIMESTEPS		TOTAL_DAYS*TIMESTEPS_PER_DAY
#define SKIP_TIMESTEPS		0


//This is the number of timesteps that the bird will skip in the beginning to get to the desired 
//takeoff time. Since the data starts at 7 pm, the birds will skip the first 23 hours to get to 
//6pm.
#define INITIAL_SKIP_TIMESTEPS		23		


//The maximum lattitude south that the model cares about bird flight. If birds go below
//that lattitude the model stops
//Counted from the North; 
#define MAX_LAT_SOUTH			300

//Stopover days; As of now, if 0 then the bird flies without stopping continiously;
//If 1, then the bird waits for 18 hours after successful 6 hours of flight to fly again
#define STOPOVER_DAYS		0

//#define DESIRED_SPEED	3.6		//Birds want to travel at 10m/s, it is 36km/hr(in the grid it is 3.6 units per hour) 
	
#define DESIRED_SPEED		10.5	//Air speed; Desired speed = flightspeed + windspeed ; Only used in windprofit calculation

#define STD_BIRDANGLE		10.0	//Standard deviation * 6 = the total difference from max to min angle possible
					//If STD_BIRDANGLE = 10 then the angle can differ +- (10*6)/2 = +- 30 from mean
#define	glCompAcc		1e-8	//If the difference is equal to or less than this then equal

#define MIN_PROFIT		-10
//Defining the x-variable size, it's sum and
//sum of squares as needed for slope calculation


#define REGRESSION_HRS		6

//Precipitation (mm/hr) below which birds can fly
#define MAX_PRECIP		2

//HRS_SUM = sum(1 to 12) before. Now has to be sum(1 to 6) = 21
#define HRS_SUM			21
#define HRS_SQUARE_SUM		91
#define DENOM_SLOPE		(REGRESSION_HRS * HRS_SQUARE_SUM)-(HRS_SUM * HRS_SUM)
// Barometric pressure
// Bird finds the pressure at the time it leaves and compares it with the data from
// the previous day.

//The angle that the bird flies when it is out at sea and needs to get back to land.
//To make the birds head back directly west the angle must be set to 180.
#define BIRD_SEA_ANGLE		180

//The maximum number of hours that the birds can fly continiously
#define BIRD_HRS_LIMIT		72

#define TOTAL_DATA_FILES	9
//Total number of data files or variables bird flight depends on;Does not include direction files and land water data
#define NUM_DATA_FILES		6

#define THREADS_PER_BLOCK	32
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

#define TOTAL_DAYS_PER_DATA_TRANSFER	5
//------------------------------Notes---------------------------------------------------------------------------------------
/*
Altitude = 850 millibars
Year = 2009
22 Jan 2015 No upper limit to the bird flight speed currently; Birds can fly well above 10m/s
Precipitation = millimeters
*/

//--------------------------------------------------------------------------------------------------------------------------
__global__ void setup_kernel(unsigned int seed,hiprandState *states,int NumOfBirds);
__global__ void generate_kernel(hiprandState *states,float* numbers,int NumOfBirds);				
__global__ void bird_movement(float* rowArray,float* colArray,int NumOfBirds,int start_l,int cur_l,int max_timesteps,float* udata,float* vdata,
				float* u10data,float* v10data,float* dirData,float* precipData,float* pressureData,
				float* lwData,uint8_t* birdStatus,int* birdTimesteps);
				
__device__ float bilinear_interpolation_SmallData(float x,float y,float* data_array);
__device__ float bilinear_interpolation_LargeData(float x,float y,float* data_array,int l);
__device__ float WrappedNormal(int id,float MeanAngle,float AngStdDev,float* rand_norm_nums,int cur_timestep);
__device__ float getProfitValue(float u_val,float v_val,float dirVal,float dir_u,float dir_v);
__device__ int bird_AtSea_Within24Hrs(int id,int arrLength,float* rowArray,float* colArray,int start_l,
					int l,float* udata,float* vdata,float* lwData,uint8_t* birdStatus,uint8_t var_product,uint8_t l_product,uint8_t l_idx);
					
__device__ float randNorm(int id, int timestep, float mean, float stdev);

static void* write_dataVars(void* arguments);
static void* read_dataFiles(void* arguments);
int convert_to_month(int month,int day);
static void HandleError( hipError_t err,const char *file, int line );
long Get_GPU_devices();
//-------------------------------------------------------------------------------------------------------------------------------------
struct file_IO {
	FILE *fp;
	float* inpVals;
	float* streamArray;
	size_t dataSize;
}inpStruct[8]; 
//-------------------------------------------------------------------------------------------------------------------------------------
//Global Variables

float* udata;
float* vdata;
float* u10data;
float* v10data;
float* precipData;
float* pressureData;

float* dir_u;
float* dir_v;
float* lwData;
float* dirData;
//-------------------------------------------------------------------------------------------------------------------------------------
__device__ __constant__ int TotalTimesteps = TIMESTEPS;
__device__ __constant__ int LatSize = LAT_SIZE;
__device__ __constant__ int LongSize = LONG_SIZE;
__device__ __constant__ float pi = PI;
__device__ __constant__ int InitialSkipTimesteps = INITIAL_SKIP_TIMESTEPS;


__device__ __constant__ int StdBirdAngle = STD_BIRDANGLE;
__device__ __constant__ int BirdSeaAngle = BIRD_SEA_ANGLE;
__device__ __constant__ int BirdHrsLimit = BIRD_HRS_LIMIT;
__device__ __constant__ int MinProfit = MIN_PROFIT;
__device__ __constant__ int MaxPrecip = MAX_PRECIP;
__device__ __constant__ int MaxLatSouth = MAX_LAT_SOUTH;
__device__ __constant__ int DesiredSpeed = DESIRED_SPEED;
__device__ __constant__ int StopoverDays = STOPOVER_DAYS;

__device__ __constant__ int DenomSlope = DENOM_SLOPE;
__device__ __constant__ int HrsSum = HRS_SUM;
__device__ __constant__ int RegressionHrs = REGRESSION_HRS;
__device__ __constant__ float GlCompAcc = glCompAcc;
__device__ __constant__ float Rand_Precision = 1000;

__device__ int CurrentTimestep = 0;

//###########################################################################################################################################//

//Getting a random normal number from Dr. David Heibler
//u1 and u2, two random numbers created using Linear Congruential Generator
//The period maximized by using Hull-Dobell Theorem (http://chagall.med.cornell.edu/BioinfoCourse/PDFs/Lecture4/random_number_generator.pdf)

//Using the theorem to get u1: a1 = 1791, m1 = 2864 and c1 = 5827. (c1 is prime therefore c1 is relatively prime with any other number; Choosing m1=2864
//	so that the prime factors are 2 and 179; Now, a1 = 179*10 + 1 as a1-1 has to be divisible by prime factors of m1)
//Using the theorem to get u2: a2 = 931, m2 = 5382 and c2 = 9461. (31, 2 and 3 are prime factors of 5382, therefore a2 = 931)
__device__ float randNorm(int id, int timestep, float mean, float stdev)
{
    int tmp, seed, a1, m1, c1, a2, m2, c2;
	float x, u1, u2;
	
	seed = (id+1)*(int)timestep;
	a1 = 1791;
	m1 = 2864;
	c1 = 5827;
	
	a2 = 931;
	m2 = 5382;
	c2 = 9461;
	
	tmp = (a1 * seed + c1) % m1;
	u1 = (float)tmp/(float)m1;
	
	tmp = (a2 * seed + c2) % m2;
	u2 = (float)tmp/(float)m2;
	
    x = sqrt(-2.0*logf(u1)) * cosf(2*pi*u2);
    x = x*stdev + mean;
	
	
    return x;
}
//###########################################################################################################################################//


__device__ int bird_AtSea_Within24Hrs(int id,int arrLength,float* rowArray,float* colArray,int start_l,int l,
float* udata,float* vdata,float* lwData,uint8_t* birdStatus,uint8_t var_product,uint8_t l_product,uint8_t l_idx)
{
	float u_val,v_val,u_dir,v_dir,pos_row,pos_col;
	float index = 0;
	int bckp_l;
	float count_timeSteps = 0;
	uint8_t var_product2;

	var_product2 = var_product;

	pos_row = rowArray[id * arrLength + l - l_idx];
	pos_col = colArray[id * arrLength + l - l_idx];

	
	
	u_dir = DesiredSpeed * cosf(BirdSeaAngle * (pi/180));
	v_dir = DesiredSpeed * sinf(BirdSeaAngle * (pi/180));

	
	for(count_timeSteps = 10;count_timeSteps<24;count_timeSteps++){
		
		var_product2 = var_product2 * birdStatus[id]; 
		
		/** Bilinear interpolation for u and v data **/
		u_val = bilinear_interpolation_LargeData(pos_col,pos_row,udata,l-start_l);	
		v_val = bilinear_interpolation_LargeData(pos_col,pos_row,vdata,l-start_l);

		/** Desired speed needs to change in the case of column position or the birds
		will not fly west **/
		pos_row = pos_row + var_product2 * (v_val + v_dir) * 0.36 * -1;	
		pos_col = pos_col + var_product2 * (u_val + u_dir) * 0.36;

		rowArray[id * arrLength + l] = pos_row;
		colArray[id * arrLength + l] = pos_col;

		//printf("At sea within 24 hours; \tRow: %f,Col:%f\n",rowArray[id * arrLength + l],colArray[id * arrLength + l]);
		//printf("At sea within 24 hours; Timestep #: %ld\n",l);

		index = lwData[__float2int_rd(pos_row * LongSize + pos_col)];
		//printf("Index after 10 hours is %f\n",index);

		if(index == 1.0){
			var_product2 = 0;		
		}else if (index == 0){ //If bird is above sea
			var_product2 = birdStatus[id];
			
		}else if (index > 1){ //If bird is above fresh water
			var_product2 = birdStatus[id];
		}
	
		if((pos_row > LatSize-1)||(pos_row >= MaxLatSouth) || (pos_col > LongSize-1)||(pos_row < 0.0)||(pos_col < 0.0 )){
			birdStatus[id] = 0;
		}

		l = l + l_product;
	}
	l = l - l_product;
	/* if(index == 0){
		birdStatus[id] = 0;
	} */
	return l;
	
}


//###########################################################################################################################################//

__device__ int bird_AtSea_After24Hrs(int id,int arrLength,float* rowArray,float* colArray,int start_l,int l,
float* udata,float* vdata,float* lwData,uint8_t* birdStatus,uint8_t var_product,uint8_t l_product)
{
	float u_val,v_val,u_dir,v_dir,pos_row,pos_col;
	int count_timeSteps, timesteps_limit, index;
	uint8_t var_product2;
		
	index = 0;
	var_product2 = var_product;

	pos_row = rowArray[id * arrLength + l - 1];
	pos_col = colArray[id * arrLength + l - 1];
	//printf("After getting the first row and cols (Inside After 24 hours function)\n");

	u_dir = DesiredSpeed * cosf(BirdSeaAngle * (pi/180));
	v_dir = DesiredSpeed * sinf(BirdSeaAngle * (pi/180));
	

	//These 25 for both condition as the for loop must 
	//be done atleast once so that l=l+l_product is done inside
	//the loop and it is offset by l=l-l_product outside the loop.
	if(l_product == 0){
		timesteps_limit = 25;
	}else{
		timesteps_limit = BirdHrsLimit;
	}

	
	if(var_product2 == 0){
		timesteps_limit = 25;
	}else{
		timesteps_limit = BirdHrsLimit;
	}

	//printf("After getting the timestep limit (Inside After 24 hours function)\n");
	//This loop is skipped if a bird is not at sea after 24 hours
	for(count_timeSteps = 24; count_timeSteps < timesteps_limit; count_timeSteps++){

		var_product2 = var_product2 * birdStatus[id]; 
		//printf("Count Timesteps:: %d\n",count_timeSteps);
		
		
		//printf("l:%ld, start_l: %ld \n",l,start_l);
		//printf("l-start_l:%ld\n",l-start_l);
		/** Bilinear interpolation for u and v data **/
		u_val = bilinear_interpolation_LargeData(pos_col,pos_row,udata,l-start_l);	
		v_val = bilinear_interpolation_LargeData(pos_col,pos_row,vdata,l-start_l);

		
		/** Desired speed needs to change in the case of column position or the birds
		will not fly west **/
		pos_row = pos_row + var_product2 * (v_val + v_dir) * 0.36 * -1;	
		pos_col = pos_col + var_product2 * (u_val + u_dir) * 0.36;

		//printf("At sea after 24 hours; \tRow: %f,Col:%f\n",rowArray[id * arrLength + l],colArray[id * arrLength + l]);
		//printf("At sea after 24 hours; Timestep #: %ld\n",l);

		rowArray[id * arrLength + l + 1] = pos_row;
		colArray[id * arrLength + l + 1] = pos_col;

		index += lwData[__float2int_rd(pos_row * LatSize + pos_col)];

		//Checking if the bird found land
		//Limit calculated only if bird found at land the first time
		if(index == 1){ //If bird is above land
			var_product2 = 0;
			timesteps_limit = __float2ull_ru(count_timeSteps/24) * 24 + 24 * StopoverDays; 					
		}else if (index == 0){ //If bird is above sea
			var_product2 = var_product2;
			
		}else if (index > 1){ //If bird is above fresh water
			var_product2 = var_product2;
		}

		//l = l + var_product2;
		l = l + l_product;

		if((pos_row > LatSize-1)||(pos_row >= MaxLatSouth) || (pos_col > LongSize-1)||(pos_row < 0.0)||(pos_col < 0.0 )){
			birdStatus[id] = 0;
		}
	}

	index = lwData[__float2int_rd(pos_row * LatSize + pos_col)];
	//printf("After getting the index (Inside After 24 hours function)\n");

	if (index != 1){
		birdStatus[id] = 0;
	}
	
	l = l - l_product;
	return l;
	
	
}

//###########################################################################################################################################//

__device__ float getProfitValue(float u_val,float v_val,float dirVal,float dir_u,float dir_v)
{

	/** All wind data in m/s **/
	float diffAngle,magnitude,magnitude_squared,tailComponent,crossComponent,profit_value;

	tailComponent = 0;
	
	magnitude = hypotf(u_val,v_val);
	magnitude_squared = magnitude * magnitude;

	/** Getting the tail component of the wind; or the component of the wind in the desired direction of flight
	From formula of getting the vector projection of wind onto the desired direction **/

	tailComponent = (dir_v * v_val + dir_u * u_val);
	tailComponent = tailComponent/hypotf(dir_u,dir_u);
	

	/** DiffAngle is the angle between the desired direction of the bird and the direction of the wind
	DiffAngle has to be calculated such that both the vectors are pointing away from where they meet.
	Using the formula to get angle between two vectors **/

	diffAngle = acosf( (u_val*dir_u + v_val * dir_v)/ (( hypotf(u_val,v_val) * hypotf(dir_u,dir_v) )) ) * 180/pi;

	/** Separate profit value methods have to be used if the tail component is less that equal to or greater than the desired speed of the birds **/
	if(tailComponent <= DesiredSpeed) {	
		profit_value = (DesiredSpeed * DesiredSpeed) + magnitude_squared - 2 * DesiredSpeed * magnitude * cosf(diffAngle * pi/180);
		profit_value = DesiredSpeed - sqrtf(profit_value);
	}
	else {
		/** Perpendicular to a vector (x,y) is (y,-x) or (-y,x) Cross component is always positive **/

		crossComponent = fabsf((-dir_v*u_val + dir_u*v_val)/hypotf(dir_v,dir_u));
		profit_value = tailComponent - crossComponent;
	}

	return profit_value;
}


//###########################################################################################################################################//

__device__ float bilinear_interpolation_SmallData(float x,float y,float* data_array)
{
	float x1,y1,x2,y2;
	float Q11,Q12,Q21,Q22,R1,R2,R;

	x1 = floorf(x);
	x2 = ceilf(x);
	y1 = floorf(y);
	y2 = ceilf(y);
	R = 0;
	
	Q11 = data_array[(int)(y1 * LongSize + x1)];
	Q12 = data_array[(int)(y2 * LongSize + x1)];
	Q21 = data_array[(int)(y1 * LongSize + x2)];
	Q22 = data_array[(int)(y2 * LongSize + x2)];
	

	R1 = Q11 + (x - x1)*(Q21 - Q11);
	R2 = Q12 + (x - x1)*(Q22 - Q12);
	R = R1 + (y - y1)*(R2 - R1);

	return R;
}


//###########################################################################################################################################//

__device__ float bilinear_interpolation_LargeData(float x,float y,float* data_array,int l)
{
	float x1,y1,x2,y2;
	float Q11,Q12,Q21,Q22,R1,R2,R;
	

	x1 = floorf(x);
	x2 = ceilf(x);
	y1 = floorf(y);
	y2 = ceilf(y);
	R = 0;
	

	Q11 = data_array[(int)(l  * LatSize * LongSize + y1 * LongSize + x1) ];
	Q12 = data_array[(int)(l  * LatSize * LongSize + y2 * LongSize + x1) ];
	Q21 = data_array[(int)(l  * LatSize * LongSize + y1 * LongSize + x2) ];
	Q22 = data_array[(int)(l  * LatSize * LongSize + y2 * LongSize + x2) ];
	

	R1 = Q11 + (x - x1)*(Q21 - Q11);
	R2 = Q12 + (x - x1)*(Q22 - Q12);
	R = R1 + (y - y1)*(R2 - R1);


	return R;
}


//###########################################################################################################################################//
/*
__global__ void setup_kernel(unsigned int seed,hiprandState *states,int NumOfBirds)
{

	//Thread indices

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int id = y * TotalTimesteps + x;


	if((x >= TotalTimesteps) || (x < 0)){
		return;
	}else if((y>= NumOfBirds) || (y < 0)){
		return;
	}else if(id >= TotalTimesteps * NumOfBirds){
		return;
	}else{
		hiprand_init(seed,id,0,&states[id]);
	}
}

//###########################################################################################################################################//

__global__ void generate_kernel(hiprandState *states,float* numbers,int NumOfBirds)
{

	//Thread indices
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int id = y * TotalTimesteps + x;

	if((x >= TotalTimesteps) || (x < 0)){
		return;
	}else if((y>= NumOfBirds) || (y < 0)){
		return;
	}else if(id >= TotalTimesteps * NumOfBirds){
		return;
	}else{
		//Making a local copy for efficiency
		hiprandState localState = states[id];	
		numbers[id] = hiprand_normal(&localState);
	}
	
	return;
}
*/
//###########################################################################################################################################//
//###########################################################################################################################################//
//###########################################################################################################################################//
__global__ void bird_movement(float* rowArray,float* colArray,int NumOfBirds,int start_l,int cur_l,int max_timesteps,float* udata,float* vdata,
				float* u10data,float* v10data,float* dirData,float* precipData,float* pressureData,
				float* lwData,uint8_t* birdStatus,int* birdTimesteps)
{

	//Thread indices
	int id = blockIdx.x * blockDim.x + threadIdx.x; 

	//if(id > (NumOfBirds -1)||(birdStatus[id]==0)||(birdTimesteps[id] > cur_l)){ 
	if(id > (NumOfBirds -1)||(birdTimesteps[id] > cur_l)){ 

	//The condition cur_l > max_timesteps is OK now because all birds start at the same time
	//NOT OK once birds start at different dates
		if(birdTimesteps[id] > cur_l){
			printf("birdTimesteps: %d, cur_l: %d\n",birdTimesteps[id],cur_l);
			
		}
		return;
	}
	
	else{


		uint8_t var_sea, var_profit_10m, var_10hrsSea, var_product, l_product,l_idx;
		//Making a local copy of the timstep variable
		int l,new_l,prev_l;
		long l_old;	
		float profit_value,actualAngle,wrappedAngle, index;
		float last_pressure,pressure_sum,pressure_MultSum,slope;
		float u_ten,v_ten,u_val,v_val,uDir_value,vDir_value,precip_val;
		int k;
		float pos_row,pos_col;
		int arrLength,days;

		//--------------Checking if timestep is larger than the current timestep
		//Should be changed from cur_l to max_timesteps
		if(birdTimesteps[id] > cur_l){
			l_product = 0;
			var_product = 0;
		}else{
			l_product = 1;
		}

	//	l_product = 1;
		l = cur_l;
		new_l = l;
		arrLength = (TotalTimesteps + 1); //Why +1 ?
	

		slope = 0;
		days = 0;
		//printf("Value of l is %ld\n",l);

		//printf("Array length per bird is %d\n",arrLength);
		//printf("id is %d\n",id);

		//printf("id * arrayLength is:%d\n",id*arrLength);
	
		//printf("Starting pos_row is %f , pos_col is: %f\n",*(rowArray + id * arrLength + l -1),*(colArray + id * arrLength + l -1));
		//printf("Before any computation; Timestep #: %ld\n",l);


		while((l < max_timesteps) && (days<5)){

			pos_row = rowArray[id * arrLength + l - 1]; //Why -1 ?
			pos_col = colArray[id * arrLength + l - 1];

			
			
			if((pos_row > LatSize-1) ||(pos_row >= MaxLatSouth) || (pos_col > LongSize-1)||(pos_row < 0.0)||(pos_col < 0.0)){
				birdStatus[id] = 0;
				printf("(Before computation) status = 0; As pos_row = %f (id:%d)\n",pos_row,id);
			}

			//--------------Getting the wrapped angle
			actualAngle = dirData[__float2int_rd(pos_row * LatSize + pos_col)];
			wrappedAngle = randNorm(id,l, actualAngle, STD_BIRDANGLE);

			if(wrappedAngle > 360){
				wrappedAngle = wrappedAngle - 360;
			
			}else if(wrappedAngle < 0 ){
				wrappedAngle = 360 + wrappedAngle;
			}	
			//--------------

			uDir_value = DesiredSpeed * cosf(wrappedAngle * (pi/180));
			vDir_value = DesiredSpeed * sinf(wrappedAngle * (pi/180));

			u_ten = bilinear_interpolation_LargeData(pos_col,pos_row,u10data,l-start_l);
			v_ten = bilinear_interpolation_LargeData(pos_col,pos_row,v10data,l-start_l);

			profit_value = getProfitValue(u_ten,v_ten,wrappedAngle,uDir_value,vDir_value);

			//--------------Checking for profit value
			if((profit_value >= MinProfit) && ((last_pressure>=1009)||(slope >-1))){
				var_profit_10m = 1;
			}else{
				var_profit_10m = 0;
				//printf("Profit value at 10m is low \n");
			}
			//--------------

		

			printf("l_product: %d \n",l_product);

			printf("Start timestep: %d\n",l);
			prev_l = l;
			
	//-----------------------------The 6 hour flight
			for(k=0;k<6;k++) {
		
				//Getting the wrapped angle
				actualAngle = dirData[__float2int_rd(pos_row * LatSize + pos_col)];
				wrappedAngle = randNorm(id,l, actualAngle, STD_BIRDANGLE);

				if(wrappedAngle > 360){
					wrappedAngle = wrappedAngle - 360;		
				}else if(wrappedAngle < 0 ){
					wrappedAngle = 360 + wrappedAngle;
				}	

				uDir_value = DesiredSpeed * cosf(wrappedAngle * (pi/180));
				vDir_value = DesiredSpeed * sinf(wrappedAngle * (pi/180));

				u_val = bilinear_interpolation_LargeData(pos_col,pos_row,udata,l-start_l); 
				v_val = bilinear_interpolation_LargeData(pos_col,pos_row,vdata,l-start_l);
				precip_val = bilinear_interpolation_LargeData(pos_col,pos_row,precipData,l-start_l);

				//Getting the previous position values for row and column
				pos_row = rowArray[id * arrLength + l - 1]; 
				pos_col = colArray[id * arrLength + l - 1];
				
				//printf("During 6 hour flight: Row: %f \t Col: %f (id:%d)\n\n\n",pos_row,pos_col,id);
			
				if((pos_row > LatSize-1)||(pos_row >= MaxLatSouth) || (pos_col > LongSize-1)||(pos_row < 0.0)||(pos_col < 0.0 )){
					birdStatus[id] = 0;
					printf("(In 6 hours flight) status = 0; As pos_row = %f (id:%d)\n",pos_row,id);
					//printf("Dead bird \n");
				}
			
				var_product = birdStatus[id] * var_profit_10m * l_product;
				
				//Storing the new values
				rowArray[id * arrLength + l] = pos_row + var_product * (v_val + vDir_value ) * 0.36 * -1;
				colArray[id * arrLength + l] = pos_col + var_product * (u_val + uDir_value) * 0.36;
					
				//printf("6 Hour Flight\tRow: %f,Col:%f\n",rowArray[id * arrLength + l],colArray[id * arrLength + l]);
				//printf("6 hour flight;Timestep #: %ld\n",l);
				
				
				l = l + l_product;

			}	
		
			//If l_product = 0 then the timestep should remain the same
			if (prev_l == l){
				l_idx = 0;				
			}else{
				l_idx = 1;
			}
			
			//The value of l increases at the last iteration 
			pos_row = rowArray[id * arrLength + l - l_idx];
			pos_col = colArray[id * arrLength + l - l_idx];
		
			index = lwData[__float2int_rd(pos_row * LatSize + pos_col)];

			// If the bird is at sea after the first 6 hours of flight 
			if( index == 1.0){
				var_sea = 0;
				printf("Not at sea after 6 hours \n");
			}else{
				var_sea = 1;
				printf("At sea after 6 hours \n");
			}

			//Getting the wrapped angle; Same uDir_value and vDir_value used for the 4 hours
			actualAngle = dirData[__float2int_rd(pos_row * LatSize + pos_col)];
			wrappedAngle = randNorm(id,l, actualAngle, STD_BIRDANGLE);
			if(wrappedAngle > 360){
				wrappedAngle = wrappedAngle - 360;
			
			}else if(wrappedAngle < 0 ){
				wrappedAngle = 360 + wrappedAngle;
			}	
			uDir_value = DesiredSpeed * cosf(wrappedAngle * (pi/180));
			vDir_value = DesiredSpeed * sinf(wrappedAngle * (pi/180));

			prev_l = l;
	//-----------------------At sea after first 6 hours of flight
			for(k=6;k<10;k++){
							
				u_val = bilinear_interpolation_LargeData(pos_col,pos_row,udata,l-start_l);
				v_val = bilinear_interpolation_LargeData(pos_col,pos_row,vdata,l-start_l);

		
		
				var_product = birdStatus[id] * var_profit_10m * var_sea * l_product;

				//Getting new position values for row and column and storing it 
				pos_row = pos_row + var_product * (v_val + vDir_value ) * 0.36 * -1;
				pos_col = pos_col + var_product * (u_val + uDir_value)  * 0.36;

				
				//printf("+4 Hour Flight\tRow: %f,Col:%f\n",pos_row,pos_col);
				//printf("+4 hour flight;Timestep #: %ld\n",l);

				if((pos_row > LatSize -1 )||(pos_row >= MaxLatSouth) || (pos_col > LongSize -1 )||(pos_row < 0.0)||(pos_col < 0.0 )){
					birdStatus[id] = 0;
					printf("(During +4 hour flight) status = 0; As pos_row = %f (id:%d)\n",pos_row,id);
				}

				rowArray[id * arrLength + l] = pos_row;

				colArray[id * arrLength + l] = pos_col;
				printf("During +4 hour flight: Row: %f \t Col: %f, u_val: %f, v_val: %f, l:%d (id:%d)\n\n\n",pos_row,pos_col,u_val,v_val,l,id);
				//printf("+4 Hour Flight\tRow: %f,Col:%f\n",rowArray[id * arrLength + l + 1],colArray[id * arrLength + l + 1]);
			
				l = l + l_product;
			}

	//------------------------

			index = lwData[__float2int_rd(pos_row * LongSize + pos_col)];


			if(index == 1){
				var_sea = 0;
				//printf("Not at sea after 10 hours \n");
			}else{
				
				var_sea = 1;
				//printf("At sea after 10 hours \n");
			}
		
	//----------------------- If at sea even after the 10 hours but within 24 hours		
			var_product = birdStatus[id] * var_profit_10m * var_sea * l_product;
			l = bird_AtSea_Within24Hrs(id,arrLength,rowArray,colArray,start_l,l,udata,vdata,lwData,birdStatus,var_product,l_product,l_idx);
	//------------------------						
			printf("Timestep after bird_AtSea_Within24Hrs %d\n",l);
			
			pos_row = rowArray[id * arrLength + l - 1]; //Why -1 ?
			pos_col = colArray[id * arrLength + l - 1];
			
			//printf("Before getting the index \n");
			index = lwData[__float2int_rd(pos_row * LongSize + pos_col)];
			if(index == 1.0){
				var_sea = 0;
				printf("Var_sea: Not at sea after 24 hours (id:%d) \n",id);
			}else{
				var_sea = 1;
				printf("Var_sea: At sea after 24 hours (id:%d) \n",id);
			}
			//printf("After getting the index \n");
	//----------------------- If at sea even after the the 10 hours and beyond 24 hours 	
			
		
			var_product = birdStatus[id] * var_profit_10m * var_sea * l_product;
			if(var_product == 1){ 
				printf("Var product = 1 : Calculations done for at sea after 24 hours(id:%d) \n",id);
			}else{
				printf("Var product = 0; No calculations for at sea after 24 hours (id:%d) \n",id);
			}
			//printf("After the variable product \n");

			printf("birdStatus[%d]: %d,var_profit_10m: %d,var_sea: %d,l_product: %d \n",id,birdStatus[id],var_profit_10m,var_sea,l_product);
			//printf("The current value of l is: %ld And of start_l is: %ld \n\n",l,start_l);
			l = bird_AtSea_After24Hrs(id,arrLength,rowArray,colArray,start_l,l,udata,vdata,lwData,birdStatus,var_product,l_product);
			printf("Timestep after bird_AtSea_After24Hrs %d (id: %d)\n",l,id);
			//days = (l - start_l)/TIMESTEPS_PER_DAY;
			//printf("Days: %d\n",days);
			//printf("After bird_AtSea_After24Hrs and before regression calculations \n");
	//------------------------	
			birdTimesteps[id] = l;
			l_old = l - RegressionHrs;

			pressure_sum = 0;
			pressure_MultSum = 0;
			
			if(birdStatus[id]==1){
				k=1;
			}else{
				k=RegressionHrs+1;
			}
			//Taking the pressure from 6 hours earlier of the location where the bird landed
			while((l_old < l) && (k<=RegressionHrs)){

				pressure_sum += bilinear_interpolation_LargeData(pos_col,pos_row,pressureData,l_old-start_l);  //<----------------ERROR HERE
				pressure_MultSum += k * bilinear_interpolation_LargeData(pos_col,pos_row,pressureData,l_old-start_l);

				//last_pressure is the last day or the day of flight
				if(k == RegressionHrs) {
					last_pressure = bilinear_interpolation_LargeData(pos_col,pos_row,pressureData,l_old-start_l);
				}
				l_old++;
				k++;
			}
			if(birdStatus[id]==1){
				slope = ((RegressionHrs * pressure_MultSum) - (pressure_sum * HrsSum))/(DenomSlope);
			}else{
				slope = 0;
			}
			
			
			printf("l: %d,birdTimesteps[id]: %d (id:%d)\n",l,birdTimesteps[id],id);
			days++;
			printf("Days: %d (id:%d)\n",days,id);
			printf("Row: %f \t Col: %f (id:%d)\n\n\n",pos_row,pos_col,id);
			printf("--------------------------------------------------------------\n");
			l = l + l_product;
					
		}
	}
}
//###########################################################################################################################################//
//###########################################################################################################################################//
//###########################################################################################################################################//
long Get_GPU_devices()
{
	hipDeviceProp_t prop;
	int whichDevice,DeviceCount;
	long deviceMemory;

	HANDLE_ERROR(hipGetDevice(&whichDevice));
	HANDLE_ERROR(hipGetDeviceProperties(&prop,whichDevice));
	
	if(!prop.deviceOverlap){
			//printf("Device does not handle overlaps so streams are not possible\n");
	return 0;
	}

	DeviceCount = 0;
	
	HANDLE_ERROR(hipGetDeviceCount(&DeviceCount));
	if(DeviceCount > 0){ 
		//printf("%d Devices Found\n",DeviceCount);
	}else{
		//printf("No devices found or error in reading the number of devices\n");
		return 0;
	}
	
	int i = 0;

	hipDeviceProp_t properties;
	HANDLE_ERROR(hipGetDeviceProperties(&properties,i));
	//printf("Device Number: %d\n", i);
	//printf("  Device name: %s\n", properties.name);
	//printf("  Device Global Memory size: %zd MB \n",properties.totalGlobalMem/1000000);
	//printf("\n");
	
	deviceMemory = properties.totalGlobalMem;


	return deviceMemory;
}

//###########################################################################################################################################//
static void* read_dataFiles(void* arguments)
{

	struct file_IO *inputArgs;
	inputArgs = (struct file_IO *)arguments;

	FILE* textFile;
	float* dataArray;


	textFile = inputArgs->fp;
	dataArray = inputArgs->inpVals;

	char line[LINESIZE];
	memset(line,'\0',sizeof(line));
	char tempVal[15];
	memset(tempVal,'\0',sizeof(tempVal));
	char* startPtr,*endPtr;
	long j;
	int i;
	float Value;
	i=0;
	j=0;

	memset(line,'\0',sizeof(line));
	memset(tempVal,'\0',sizeof(tempVal));
	i=0;
	j=0;

	while(fgets(line,LINESIZE,textFile)!=NULL){
		startPtr = line;
		for(i=0;i<LONG_SIZE;i++){
			Value = 0;
			memset(tempVal,'\0',sizeof(tempVal));

			if(i != (LONG_SIZE - 1)) {
				endPtr = strchr(startPtr,',');
				strncpy(tempVal,startPtr,endPtr-startPtr);
				//printf("%s ",tempVal);
				if(strcmp("NaN",tempVal)==0) {	
					Value = 0.0;
					
				}
				else{
					Value = atof(tempVal);
				
				}
				
				dataArray[j * LAT_SIZE + i] = Value;
				endPtr = endPtr + 1;
				startPtr = endPtr;
				//printf("%d,%f ",i,Value);
			}
			else if(i == (LONG_SIZE - 1)){
				strcpy(tempVal,startPtr);

				if(strcmp("NaN\n",tempVal)==0) {
					Value = 0.0;
				}
				else{
					Value = atof(tempVal);
				}
				dataArray[j * LAT_SIZE + i] = Value;
			}
		}
		j++;
	}
	return NULL;
}

//###########################################################################################################################################//
static void* write_dataVars(void* arguments)
{

	struct file_IO *inputArgs;
	inputArgs = (struct file_IO *)arguments;

	float* dataArray,*destArray;
	size_t totalSize;	
	long int i;

	dataArray = inputArgs->inpVals;
	destArray = inputArgs->streamArray;
	totalSize = inputArgs->dataSize;

	for(i=0;i<totalSize;i++){
		destArray[i] = *(dataArray + i);
	}

	return NULL;
}



//###########################################################################################################################################//
int convert_to_month(int month,int day)
{
	int index,offset;
	if(month == 8){
		index = 1; //The data starts in august
	}
	else if(month == 9){
		index = 32; //The data for september starts after 31 days of august
	}
	else if(month == 10){
		index = 62; //The data for october starts after 31+30 days of sept and august respectively.
	}
	else if(month == 11){
		index = 93; //The data for october starts after 31+30+31 days of sept,aug and oct respectively.
	}
	else{
		//printf("\n\t\tIncorrect month used\n\t\tUse between August-November inclusive; Only use numbers ; August = 8\n");
		return -1;
	}

	//If 1st or 2nd of August, start at timestep 23 (after 23 hours)
	if(((month == 8) && (day == 1))||((month == 8) && (day == 2))){
		offset = 23;
	//If in August; Gives correct result for starting timestep
	}else if (month == 8){
		offset = 23 + (day - 1) * TIMESTEPS_PER_DAY ;
	//23 added because 1st day only has 23 hours
	}else{
		offset = 23 + (index - 2 + day) * TIMESTEPS_PER_DAY;
	}

	return offset;

}

//###########################################################################################################################################//

static void HandleError( hipError_t err,const char *file, int line ) {
    if (err != hipSuccess) {
  		printf( "%s in %s at line %d\n", hipGetErrorString( err ),file, line );
        exit( EXIT_FAILURE );
    }
}

//###########################################################################################################################################//
//###########################################################################################################################################//
//###########################################################################################################################################//


int main(int argc,char* argv[])
{

//--------------------------Checking for input arguments------------------------------//

	char baseFileName[] = "../../Birds_Full/Birds_data/InterpolatedData/";
	char yearFileName[80];
	char fullFileName[80];
	char start_date[12];
	char yearStr[4],monthStr[2],dayStr[2];

	float starting_row,starting_col;
	int offset_into_data = 0;
	int NumOfBirds,year,day,month;

	int option;
	
	while ((option = getopt(argc, argv,"y:m:d:r:c:N:")) != -1) {
        	switch (option) {
             		case 'y' : year = atoi(optarg);
             		    break;
             		case 'm' : month = atoi(optarg);
             		    break;
             		case 'd' : day = atoi(optarg); 
             		    break;
             		case 'r' : starting_row = atof(optarg);
             		    break;
             		case 'c' : starting_col = atof(optarg);
             		    break;
             //		case 't' : breadth = atoi(optarg);
             //		    break;
             		case 'N' : NumOfBirds = atoi(optarg);
             		    break;
             		default: printf("\nUsage: birds -y Year -m Month(Number) -d DayOfTheMonth -r StartingRow -c StartingCol -N NumberOfBirds\n"); 
             		    exit(EXIT_FAILURE);
        	}
   	 }

	
	/** If starting row is greater than or equal the row that we are interested in; Below a particular row we are not interested in the flight of the birds**/
	if(starting_row >= MAX_LAT_SOUTH){
		printf("\t\tProvided starting row is below the southern most lattitude at which the model is set to stop\n");
		printf("\t\tEither change the starting row location and/or MAX_LAT upto which the birds can fly\n");
		exit(-1);
	}
	
//-----------------------------------------------Day-----------------------------------------//
/** Making sure random date is not provided **/

	if((day>0) && (day<32)){
		sprintf(dayStr,"%d",day);
	}else{
		printf("\t\t Invalid date provided; Date should be greater than 0 and less than 32\n");
		exit(-1);
	}

//-----------------------------------------------Month-----------------------------------------//
/** Making sure month provided is between August and November inclusive **/

	if((month < 12) && (month > 7)){
		sprintf(monthStr,"%d",month);
	}else{
		printf("\t\t Invalid month provided; Use between 8 and 11 inclusive\n");
		exit(-1);
	}

	/** Converting month and day information into number of timesteps; Special case of AUG 1st is also taken care of
	Instead of AUG 1 it starts at August 2 (because data starts at 7pm but birds fly at 6pm) **/
	offset_into_data = convert_to_month(month,day);
	
	printf("Offset into data is: %d\n",offset_into_data);

//-----------------------------------------------Year-----------------------------------------//
/** Checking if correct year specified **/

	if((year>= 2008) && (year<=2013)){
		//Add file location here
		sprintf(yearStr,"%d",year);
		strcpy(yearFileName,baseFileName);
		strcat(yearFileName,yearStr);
		strcat(yearFileName,"/");
	}
	else{
		printf("\n\tInvalid year specified\n\tSpecified %d; Use years from 2008 to 2013 in its full format\n",year);
             	printf("\t\tUsage: birds -y Year -m Month(Number) -d DayOfTheMonth -r StartingRow -c StartingCol -N NumberOfBirds\n"); 
		exit(-1);		
	}

	strcpy(start_date,yearStr);
	strcat(start_date,"/");	
	strcat(start_date,monthStr);
	strcat(start_date,"/");
	sprintf(dayStr,"%d",day);
	strcat(start_date,dayStr);

//------------Opening row and column output data file where lat and long
//-------------------------------------------- positions are stored--------//
	FILE *rowdataTxt,*coldataTxt,*birdStatusTxt;
	FILE *vdataTxt,*udataTxt,*v10dataTxt,*u10dataTxt,*precipTxt,*pressureTxt,*lwTxt,*dirTxt;

	rowdataTxt = fopen("./Output/row_output.txt","a");
	if(rowdataTxt == NULL) {
		perror("Cannot open output row data file\n");
		exit(-1);
	}

	coldataTxt = fopen("./Output/col_output.txt","a");
	if(coldataTxt == NULL) {
		perror("Cannot open output col data file\n");
		exit(-1);
	}

	birdStatusTxt = fopen("./Output/birdStatus_Final.txt","a");
	if(birdStatusTxt == NULL) {
		perror("Cannot open output birdStatus file\n");
		exit(-1);
	}
//----------------------Opening U850 data file----------------------------//
	memset(fullFileName,0,strlen(fullFileName));
	strcpy(fullFileName,yearFileName);
	strcat(fullFileName,"U850.txt");

	printf("U50 filename is %s \n",fullFileName);
	udataTxt = fopen(fullFileName,"r");

	if(udataTxt == NULL) {
		perror("Cannot open file with U850 data\n");
		exit(-1);
	}
//------------------------Opening V850 data file--------------------------//
	memset(fullFileName,0,strlen(fullFileName));
	strcpy(fullFileName,yearFileName);
	strcat(fullFileName,"V850.txt");

	vdataTxt = fopen(fullFileName,"r");

	if(vdataTxt == NULL) {
		perror("Cannot open file with V850 data\n");
		exit(-1);
	}
//-----------------------Opening U10 data file---------------------------//
	//Birds will check the wind at the surface therefore the u and v
	//at 10m is required
	
	memset(fullFileName,0,strlen(fullFileName));
	strcpy(fullFileName,yearFileName);
	strcat(fullFileName,"U10.txt");

	u10dataTxt = fopen(fullFileName,"r");

	if(u10dataTxt == NULL) {
		perror("Cannot open file with U10 data\n");
		exit(-1);
	}
//-----------------------Opening V10 data file---------------------------//
	memset(fullFileName,0,strlen(fullFileName));
	strcpy(fullFileName,yearFileName);
	strcat(fullFileName,"V10.txt");

	v10dataTxt = fopen(fullFileName,"r");
	
	if(v10dataTxt == NULL) {
		perror("Cannot open file with V10 data\n");
		exit(-1);
	}
//--------------------Opening PRCP data file------------------------------//
	memset(fullFileName,0,strlen(fullFileName));
	strcpy(fullFileName,yearFileName);
	strcat(fullFileName,"PRCP.txt");

	precipTxt = fopen(fullFileName,"r");
	if(precipTxt == NULL) {
		perror("Cannot open file with PRCP data\n");
		exit(-1);
	}
//------------------------Opening MSLP data file--------------------------//
	memset(fullFileName,0,strlen(fullFileName));
	strcpy(fullFileName,yearFileName);
	strcat(fullFileName,"MSLP.txt");

	pressureTxt = fopen(fullFileName,"r");
	if(pressureTxt == NULL) {
		perror("Cannot open file with pressure data!\n");
		exit(-1);
	}
//--------------------------Opening Land vs Water File---------------------//
	lwTxt = fopen("./Lw_and_Dir/land_water_detail.txt","r");
	if(lwTxt == NULL) {
		perror("Cannot open file with direction data\n");
		exit(-1);
	}
//--------------------------Opening Direction file 
//--------------------(Example: ext_crop.txt or extP_crop.txt)-------------//

	dirTxt = fopen("./Lw_and_Dir/ext_Final_NewCoordSystem.txt","r");
	//dirTxt = fopen("ext_crop.txt","r");
	if(dirTxt == NULL) {
		perror("Cannot open file with direction data\n");
		exit(-1);
	}


//-----------------------------Setting Heap Size,printf buffer size etc--------------------------------------------//
//	size_t limit;
//	HANDLE_ERROR(hipDeviceSetLimit(hipLimitPrintfFifoSize, 500 * 1024 * 1024));
//	hipDeviceGetLimit(&limit,hipLimitPrintfFifoSize);


//	HANDLE_ERROR(hipDeviceSetLimit(hipLimitMallocHeapSize,(size_t)(6 * LAT_SIZE * LONG_SIZE * TIMESTEPS * sizeof(float))));
//--------------------------Memory Allocation for global arrays containing weather data----------------------------//
	HANDLE_ERROR(hipDeviceReset());

	float *h_row,*h_col;
	float *d_row,*d_col;	
	float *d_udata,*d_vdata,*d_u10data,*d_v10data,*d_lwData;
	float *d_dirData,*d_precipData,*d_pressureData;
	int *h_birdTimesteps, *d_birdTimesteps;
	uint8_t *h_birdStatus,*d_birdStatus;

	//Pinned memory is faster than non-pinned memory only if the amount of transferred data
	//is above 16GB 
	// https://www.cs.virginia.edu/~mwb7w/cuda_support/pinned_tradeoff.html

	dirData = (float*) malloc(LAT_SIZE * LONG_SIZE * sizeof(float));
	h_row = (float*) malloc(NumOfBirds * (TIMESTEPS + 1) * sizeof(float));
	h_col = (float*) malloc(NumOfBirds * (TIMESTEPS + 1) * sizeof(float));
	h_birdStatus = (uint8_t*)malloc(NumOfBirds * sizeof(uint8_t));
	h_birdTimesteps = (int*)malloc(NumOfBirds * sizeof(int));
	lwData = (float*) malloc(LAT_SIZE * LONG_SIZE * sizeof(float));	

//------------------------------------------------------------------------------------------------------------------//

	HANDLE_ERROR(hipHostMalloc((void **)&udata,LAT_SIZE * LONG_SIZE * TIMESTEPS * sizeof(float)));
	HANDLE_ERROR(hipHostMalloc((void **)&vdata,LAT_SIZE * LONG_SIZE * TIMESTEPS * sizeof(float)));
	HANDLE_ERROR(hipHostMalloc((void **)&u10data,LAT_SIZE * LONG_SIZE * TIMESTEPS * sizeof(float)));	
	HANDLE_ERROR(hipHostMalloc((void **)&v10data,LAT_SIZE * LONG_SIZE * TIMESTEPS * sizeof(float)));
	HANDLE_ERROR(hipHostMalloc((void **)&precipData,LAT_SIZE * LONG_SIZE * TIMESTEPS * sizeof(float)));
	HANDLE_ERROR(hipHostMalloc((void **)&pressureData,LAT_SIZE * LONG_SIZE * TIMESTEPS * sizeof(float)));

	//printf("Size of large arrays is %zd\n",sizeof(udata)/sizeof(udata[0]));
	//printf("Size of large arrays is %ld\n",sizeof(udata)/sizeof(float));
	//printf("Size of large arrays is %ld\n",sizeof(udata)/sizeof(float));

	int ii,jj;
	for(ii=0;ii<(NumOfBirds * (TIMESTEPS + 1));ii++){
		*(h_row + ii) = starting_row;
		*(h_col + ii) = starting_col;
	}

	//Setting the current status of the birds to Alive
	//And the current timestep at the starting timestep defined by the user
	for(ii=0;ii<NumOfBirds;ii++){
		h_birdStatus[ii] = (uint8_t)1;
		h_birdTimesteps[ii] = (int)offset_into_data;
	}

//--------------------------Initializing the structures for pthreads-----------------------------------------------------------//

	
	inpStruct[0].fp = vdataTxt;
	inpStruct[0].inpVals = vdata;

	inpStruct[1].fp = udataTxt;
	inpStruct[1].inpVals = udata;

	inpStruct[2].fp = v10dataTxt;
	inpStruct[2].inpVals = v10data;

	inpStruct[3].fp = u10dataTxt;
	inpStruct[3].inpVals = u10data;

	inpStruct[4].fp = precipTxt;
	inpStruct[4].inpVals = precipData;

	inpStruct[5].fp = pressureTxt;
	inpStruct[5].inpVals = pressureData;

	inpStruct[6].fp = lwTxt;
	inpStruct[6].inpVals = lwData;

	inpStruct[7].fp = dirTxt;
	inpStruct[7].inpVals = dirData;


	/** Using pthreads to read from the files in parallel**/
	pthread_t threads[8];


	printf("Before pthreads creation \n");
	int i;
	for(i=0;i<8;i++){
		if(pthread_create(&threads[i],NULL,read_dataFiles,(void*)&inpStruct[i]) != 0){
			fprintf(stderr,"ERROR: Thread creation using pthreads failed\n");
			exit(-1);
		}

	}
	printf("After pthreads creation and before join\n ");
	for(i=0;i<8;i++){
		if(pthread_join(threads[i],NULL)!=0){
 			fprintf(stderr,"ERROR: Thread join failed\n");
                        exit(-1);
		}
	}
	printf("After pthreads join\n ");

	printf("End of parallel data read\n");
	

	int DeviceCount;
	/** Getting the total number of devices available **/
	HANDLE_ERROR(hipGetDeviceCount(&DeviceCount));
	HANDLE_ERROR(hipSetDevice(DeviceCount - 1));
	

//-------------------------------------------------------------------------------------------------------------------------//	
	HANDLE_ERROR(hipMalloc((void**)&d_row,NumOfBirds * (TIMESTEPS + 1 ) * sizeof(float)));	
	HANDLE_ERROR(hipMalloc((void**)&d_col,NumOfBirds * (TIMESTEPS + 1 ) * sizeof(float)));	
	HANDLE_ERROR(hipMalloc((void**)&d_lwData,LAT_SIZE * LONG_SIZE * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&d_dirData,LAT_SIZE * LONG_SIZE * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&d_birdStatus,NumOfBirds * sizeof(uint8_t)));
	HANDLE_ERROR(hipMalloc((void**)&d_birdTimesteps,NumOfBirds * sizeof(int)));


	printf("After hipMalloc of non changing data\n");

/*
	HANDLE_ERROR(hipMemcpyAsync(d_row,h_row,NumOfBirds * (TIMESTEPS + 1 ) * sizeof(float),hipMemcpyHostToDevice,streams_posData[0]));
	HANDLE_ERROR(hipMemcpyAsync(d_col,h_col,NumOfBirds * (TIMESTEPS + 1 ) * sizeof(float),hipMemcpyHostToDevice,streams_posData[1]));
	HANDLE_ERROR(hipMemcpyAsync(d_lwData,lwData,LAT_SIZE * LONG_SIZE * sizeof(float),hipMemcpyHostToDevice,streams_posData[2]));
	HANDLE_ERROR(hipMemcpyAsync(d_dirData,dirData,LAT_SIZE * LONG_SIZE * sizeof(float),hipMemcpyHostToDevice,streams_posData[3]));
	HANDLE_ERROR(hipMemcpyAsync(d_birdStatus,h_birdStatus,NumOfBirds * sizeof(uint8_t),hipMemcpyHostToDevice,streams_posData[4]));
*/

	HANDLE_ERROR(hipMemcpy(d_row,h_row,NumOfBirds * (TIMESTEPS + 1 ) * sizeof(float),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_col,h_col,NumOfBirds * (TIMESTEPS + 1 ) * sizeof(float),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_lwData,lwData,LAT_SIZE * LONG_SIZE * sizeof(float),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_dirData,dirData,LAT_SIZE * LONG_SIZE * sizeof(float),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_birdStatus,h_birdStatus,NumOfBirds * sizeof(uint8_t),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_birdTimesteps,h_birdTimesteps,NumOfBirds * sizeof(int),hipMemcpyHostToDevice));

	printf("After hipMemcpy of non changing data\n");
//------------------------------------Getting the size of data needed per transfer---------------------------------------------//

	//Maximum number of days that a bird can fly continiously
	int MaxFlightDays = BIRD_HRS_LIMIT/TIMESTEPS_PER_DAY;
	long int TimestepsPerTransfer = (TOTAL_DAYS_PER_DATA_TRANSFER + MaxFlightDays) * TIMESTEPS_PER_DAY;
	long int TotalDataPerIteration = TimestepsPerTransfer * LAT_SIZE * LONG_SIZE * sizeof(float);
	long int TotalDataPerDay = TIMESTEPS_PER_DAY * LAT_SIZE * LONG_SIZE * sizeof(float);
	long int TotalDataForThreeDays = 3 * TIMESTEPS_PER_DAY * LAT_SIZE * LONG_SIZE * sizeof(float);

//	int DaysTransferrable = TOTAL_DAYS - 1 - (MaxFlightDays - 1);
	int DaysTransferrable = ((TOTAL_DAYS - 1 - MaxFlightDays + 1)/TOTAL_DAYS_PER_DATA_TRANSFER) * TOTAL_DAYS_PER_DATA_TRANSFER;
	int DaysRemaining_Transferrable = (TOTAL_DAYS - MaxFlightDays) - DaysTransferrable ;

//-----------------------------------------------------------------------------------------------------------------------------//
	long int h_offset,d_offset,h_offsetStart,d_offsetStart;
	int start_timestep,cur_timestep,max_timesteps;
	
	h_offset = INITIAL_SKIP_TIMESTEPS * TIMESTEPS_PER_DAY * LAT_SIZE * LONG_SIZE;
	d_offset = 0;
	cur_timestep = offset_into_data;

	HANDLE_ERROR(hipSetDevice(DeviceCount - 1));

	printf("After selecting the correct device\n");
	HANDLE_ERROR(hipMalloc((void**)&d_udata,TotalDataPerIteration));	
	HANDLE_ERROR(hipMalloc((void**)&d_vdata,TotalDataPerIteration));	
	HANDLE_ERROR(hipMalloc((void**)&d_u10data,TotalDataPerIteration));	
	HANDLE_ERROR(hipMalloc((void**)&d_v10data,TotalDataPerIteration));	
	HANDLE_ERROR(hipMalloc((void**)&d_precipData,TotalDataPerIteration));	
	HANDLE_ERROR(hipMalloc((void**)&d_pressureData,TotalDataPerIteration));

	printf("After hipMalloc for the changing data\n");
	HANDLE_ERROR(hipDeviceSynchronize());

	int current_index, next_index;

	//current_index = ((offset_into_data - INITIAL_SKIP_TIMESTEPS)/24) % total_streams;
	ii = -1;
	jj = 0;
	current_index = 0;

	//printf("After hipMemcpyAsync for the changing data\n");

	dim3 gridSize((NumOfBirds + 32 - 1)/32,1,1);
	dim3 blockSize(32,1,1);

	int zz = 0;

//-----------------------------------Creating streams-------------------------------------------//
	//Hardcoded for Kepler architecture
	const int total_streams = 32;

	hipStream_t streams[total_streams];

	for(i = 0;i<total_streams;i++){
		HANDLE_ERROR(hipStreamCreate(&streams[i]));
	}

	printf("After streams creation for the changing data\n");


	for(i=0;i<DaysTransferrable;i=i+1){

		//HANDLE_ERROR(hipSetDevice(DeviceCount - 1));
		start_timestep = i * TIMESTEPS_PER_DAY + INITIAL_SKIP_TIMESTEPS;
		max_timesteps = start_timestep + TimestepsPerTransfer;
		cur_timestep = start_timestep;

		//Has to change once the start dates for each bird changes
		if(start_timestep >= offset_into_data){

			//All of these are inclusive
			//If TimeStepsPerTransfer is 9, then they would be: 0-8, 9-17, 18-26,...
			start_timestep = i * TIMESTEPS_PER_DAY + INITIAL_SKIP_TIMESTEPS;
			max_timesteps = start_timestep + TimestepsPerTransfer;
			cur_timestep = start_timestep;

			ii = ii + 1;
			h_offset = (TIMESTEPS_PER_DAY * LAT_SIZE * LONG_SIZE) * i  + INITIAL_SKIP_TIMESTEPS;
			d_offset = TotalDataPerDay/sizeof(float) *(ii % 5);

			h_offsetStart = h_offset;
			d_offsetStart = d_offset;

			HANDLE_ERROR(hipMemcpyAsync(d_udata + d_offset,udata+h_offset,TotalDataPerDay,hipMemcpyHostToDevice,streams[0]));
			HANDLE_ERROR(hipMemcpyAsync(d_vdata + d_offset,vdata+h_offset,TotalDataPerDay,hipMemcpyHostToDevice,streams[1]));
			HANDLE_ERROR(hipMemcpyAsync(d_u10data + d_offset,u10data+h_offset,TotalDataPerDay,hipMemcpyHostToDevice,streams[2]));
			HANDLE_ERROR(hipMemcpyAsync(d_v10data + d_offset,v10data+h_offset,TotalDataPerDay,hipMemcpyHostToDevice,streams[3]));
			HANDLE_ERROR(hipMemcpyAsync(d_precipData + d_offset,precipData+h_offset,TotalDataPerDay,hipMemcpyHostToDevice,streams[4]));
			HANDLE_ERROR(hipMemcpyAsync(d_pressureData + d_offset,pressureData+h_offset,TotalDataPerDay,hipMemcpyHostToDevice,streams[5]));
			i = i + 1;
			//-----------------------------------------------------------------------------------------------------------------------------//
			ii = ii + 1;
			h_offset = (TIMESTEPS_PER_DAY * LAT_SIZE * LONG_SIZE) * i  + INITIAL_SKIP_TIMESTEPS;
			d_offset = TotalDataPerDay/sizeof(float) *(ii % 5);

			HANDLE_ERROR(hipMemcpyAsync(d_udata + d_offset,udata+h_offset,TotalDataPerDay,hipMemcpyHostToDevice,streams[6]));
			HANDLE_ERROR(hipMemcpyAsync(d_vdata + d_offset,vdata+h_offset,TotalDataPerDay,hipMemcpyHostToDevice,streams[7]));
			HANDLE_ERROR(hipMemcpyAsync(d_u10data + d_offset,u10data+h_offset,TotalDataPerDay,hipMemcpyHostToDevice,streams[8]));
			HANDLE_ERROR(hipMemcpyAsync(d_v10data + d_offset,v10data+h_offset,TotalDataPerDay,hipMemcpyHostToDevice,streams[9]));
			HANDLE_ERROR(hipMemcpyAsync(d_precipData + d_offset,precipData+h_offset,TotalDataPerDay,hipMemcpyHostToDevice,streams[10]));
			HANDLE_ERROR(hipMemcpyAsync(d_pressureData + d_offset,pressureData+h_offset,TotalDataPerDay,hipMemcpyHostToDevice,streams[11]));
			i = i + 1;
			//-----------------------------------------------------------------------------------------------------------------------------//
			ii = ii + 1;
			h_offset = (TIMESTEPS_PER_DAY * LAT_SIZE * LONG_SIZE) * i + INITIAL_SKIP_TIMESTEPS;
			d_offset = TotalDataPerDay/sizeof(float) *(ii % 5);

			HANDLE_ERROR(hipMemcpyAsync(d_udata + d_offset,udata+h_offset,TotalDataPerDay,hipMemcpyHostToDevice,streams[12]));
			HANDLE_ERROR(hipMemcpyAsync(d_vdata + d_offset,vdata+h_offset,TotalDataPerDay,hipMemcpyHostToDevice,streams[13]));
			HANDLE_ERROR(hipMemcpyAsync(d_u10data + d_offset,u10data+h_offset,TotalDataPerDay,hipMemcpyHostToDevice,streams[14]));
			HANDLE_ERROR(hipMemcpyAsync(d_v10data + d_offset,v10data+h_offset,TotalDataPerDay,hipMemcpyHostToDevice,streams[15]));
			HANDLE_ERROR(hipMemcpyAsync(d_precipData + d_offset,precipData+h_offset,TotalDataPerDay,hipMemcpyHostToDevice,streams[16]));
			HANDLE_ERROR(hipMemcpyAsync(d_pressureData + d_offset,pressureData+h_offset,TotalDataPerDay,hipMemcpyHostToDevice,streams[17]));
			i = i + 1;
			//-----------------------------------------------------------------------------------------------------------------------------//
			ii = ii + 1;
			h_offset = (TIMESTEPS_PER_DAY * LAT_SIZE * LONG_SIZE) * i + INITIAL_SKIP_TIMESTEPS;
			d_offset = TotalDataPerDay/sizeof(float) *(ii % 5);

			HANDLE_ERROR(hipMemcpyAsync(d_udata + d_offset,udata+h_offset,TotalDataPerDay,hipMemcpyHostToDevice,streams[18]));
			HANDLE_ERROR(hipMemcpyAsync(d_vdata + d_offset,vdata+h_offset,TotalDataPerDay,hipMemcpyHostToDevice,streams[19]));
			HANDLE_ERROR(hipMemcpyAsync(d_u10data + d_offset,u10data+h_offset,TotalDataPerDay,hipMemcpyHostToDevice,streams[20]));
			HANDLE_ERROR(hipMemcpyAsync(d_v10data + d_offset,v10data+h_offset,TotalDataPerDay,hipMemcpyHostToDevice,streams[21]));
			HANDLE_ERROR(hipMemcpyAsync(d_precipData + d_offset,precipData+h_offset,TotalDataPerDay,hipMemcpyHostToDevice,streams[22]));
			HANDLE_ERROR(hipMemcpyAsync(d_pressureData + d_offset,pressureData+h_offset,TotalDataPerDay,hipMemcpyHostToDevice,streams[23]));
			i = i + 1;
			//-----------------------------------------------------------------------------------------------------------------------------//
			ii = ii + 1;
			h_offset = (TIMESTEPS_PER_DAY * LAT_SIZE * LONG_SIZE) * i + INITIAL_SKIP_TIMESTEPS;
			d_offset = TotalDataPerDay/sizeof(float) *(ii % 5);

			HANDLE_ERROR(hipMemcpyAsync(d_udata + d_offset,udata+h_offset,TotalDataForThreeDays,hipMemcpyHostToDevice,streams[24]));
			HANDLE_ERROR(hipMemcpyAsync(d_vdata + d_offset,vdata+h_offset,TotalDataForThreeDays,hipMemcpyHostToDevice,streams[25]));
			HANDLE_ERROR(hipMemcpyAsync(d_u10data + d_offset,u10data+h_offset,TotalDataForThreeDays,hipMemcpyHostToDevice,streams[26]));
			HANDLE_ERROR(hipMemcpyAsync(d_v10data + d_offset,v10data+h_offset,TotalDataForThreeDays,hipMemcpyHostToDevice,streams[27]));
			HANDLE_ERROR(hipMemcpyAsync(d_precipData + d_offset,precipData+h_offset,TotalDataForThreeDays,hipMemcpyHostToDevice,streams[28]));
			HANDLE_ERROR(hipMemcpyAsync(d_pressureData + d_offset,pressureData+h_offset,TotalDataForThreeDays,hipMemcpyHostToDevice,streams[29]));

			for(jj=0;jj<total_streams;jj++){
			 	HANDLE_ERROR(hipStreamSynchronize(streams[jj]));
			}
			
			printf("########################################################################");
			printf("Kernel call# %d\n",zz);
			
			bird_movement<<<gridSize,blockSize,0,streams[30]>>>(d_row,d_col,NumOfBirds,start_timestep,cur_timestep,max_timesteps,
			d_udata,d_vdata,d_u10data,d_v10data,d_dirData,d_precipData,d_pressureData,d_lwData,d_birdStatus,d_birdTimesteps);

			zz++;
			
			HANDLE_ERROR(hipStreamSynchronize(streams[30]));


		}

	}

	
	printf("Number of days: %d\n",i);

	HANDLE_ERROR(hipMemcpy(h_row,d_row,NumOfBirds * (TIMESTEPS + 1 ) * sizeof(float),hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(h_col,d_col,NumOfBirds * (TIMESTEPS + 1 ) * sizeof(float),hipMemcpyDeviceToHost));
	
	for(i = 0;i < NumOfBirds * (TIMESTEPS + 1); i++ ){
		fprintf(rowdataTxt,"%f ",h_row[i]);	
		if(((i+1) % (TIMESTEPS + 1)) == 0){
			fprintf(rowdataTxt,"%f \n",h_row[i]);
		}

	}

	for(i = 0;i < NumOfBirds * (TIMESTEPS + 1); i++ ){
		fprintf(coldataTxt,"%f ",h_col[i]);	
		if(((i+1) % (TIMESTEPS + 1)) == 0){
			fprintf(coldataTxt,"%f \n",h_col[i]);
		}

	}

//-----------------------------------------------Freeing allocated memory--------------------------------------//
	for(i = 0;i<total_streams;i++){
		HANDLE_ERROR(hipStreamDestroy(streams[i]));
	}



	HANDLE_ERROR(hipFree(d_udata));
	HANDLE_ERROR(hipFree(d_vdata));
	HANDLE_ERROR(hipFree(d_u10data));
	HANDLE_ERROR(hipFree(d_v10data));
	HANDLE_ERROR(hipFree(d_precipData));
	HANDLE_ERROR(hipFree(d_pressureData));

	HANDLE_ERROR(hipFree(d_row));	
	HANDLE_ERROR(hipFree(d_col));	
	HANDLE_ERROR(hipFree(d_birdStatus));
	HANDLE_ERROR(hipFree(d_birdTimesteps));		


	
	HANDLE_ERROR(hipHostFree(udata));
	HANDLE_ERROR(hipHostFree(vdata));
	HANDLE_ERROR(hipHostFree(u10data));
	HANDLE_ERROR(hipHostFree(v10data));
	HANDLE_ERROR(hipHostFree(precipData));
	HANDLE_ERROR(hipHostFree(pressureData));


	free(dirData);
	free(h_row);
	free(h_col);
	free(lwData);
	free(h_birdStatus);
	free(h_birdTimesteps);
	
	fclose(birdStatusTxt);
	fclose(dirTxt);
	fclose(udataTxt);
	fclose(vdataTxt);
	fclose(v10dataTxt);
	fclose(u10dataTxt);
	fclose(precipTxt);
	fclose(pressureTxt);
	fclose(lwTxt);
	fclose(rowdataTxt);
	fclose(coldataTxt);
	
	//printf("End\n");
	HANDLE_ERROR(hipDeviceReset());
	exit(0);
	return 0;
}








