#include "hip/hip_runtime.h"

//Needs Header Files for the functions; The header file should have both C and CUDA functions



//This file uses 6 hourly data. Each day is 6 hours long and skipping a day means to add 6
//to the counter that counts the timesteps (l).

//The birds start at 00:00 UTC which is 6pm in central time examplewhen there is no day light savings
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>

#include <pthread.h>
#include <string.h>
#include <math.h>
#include <float.h>

#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <getopt.h>

#include <math.h>


//#include "birds_CUDA.h"
//#define CUDA_API_PER_THREAD_DEFAULT_STREAM


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>


#define PI 			3.14159
#define LONG_SIZE		429
#define LAT_SIZE		429
#define LINESIZE		15*LONG_SIZE+LONG_SIZE - 3
#define TOTAL_DAYS		122
#define TIMESTEPS_PER_DAY	24
#define TIMESTEPS		TOTAL_DAYS*TIMESTEPS_PER_DAY
#define SKIP_TIMESTEPS		0


//This is the number of timesteps that the bird will skip in the beginning to get to the desired 
//takeoff time. Since the data starts at 7 pm, the birds will skip the first 23 hours to get to 
//6pm.
#define INITIAL_SKIP_TIMESTEPS		23		


//The maximum lattitude south that the model cares about bird flight. If birds go below
//that lattitude the model stops
//Counted from the North; 
#define MAX_LAT_SOUTH			300

//Stopover days; As of now, if 0 then the bird flies without stopping continiously;
//If 1, then the bird waits for 18 hours after successful 6 hours of flight to fly again
#define STOPOVER_DAYS		0

//#define DESIRED_SPEED	3.6		//Birds want to travel at 10m/s, it is 36km/hr(in the grid it is 3.6 units per hour) 
	
#define DESIRED_SPEED		10.5	//Air speed; Desired speed = flightspeed + windspeed ; Only used in windprofit calculation

#define STD_BIRDANGLE		10.0	//Standard deviation * 6 = the total difference from max to min angle possible
					//If STD_BIRDANGLE = 10 then the angle can differ +- (10*6)/2 = +- 30 from mean
#define	glCompAcc		1e-8	//If the difference is equal to or less than this then equal

#define MIN_PROFIT		-10
//Defining the x-variable size, it's sum and
//sum of squares as needed for slope calculation


#define REGRESSION_HRS		6

//Precipitation (mm/hr) below which birds can fly
#define MAX_PRECIP		2

//HRS_SUM = sum(1 to 12) before. Now has to be sum(1 to 6) = 21
#define HRS_SUM			21
#define HRS_SQUARE_SUM		91
#define DENOM_SLOPE		(REGRESSION_HRS * HRS_SQUARE_SUM)-(HRS_SUM * HRS_SUM)
// Barometric pressure
// Bird finds the pressure at the time it leaves and compares it with the data from
// the previous day.

//The angle that the bird flies when it is out at sea and needs to get back to land.
//To make the birds head back directly west the angle must be set to 180.
#define BIRD_SEA_ANGLE		180

//The maximum number of hours that the birds can fly continiously
#define BIRD_HRS_LIMIT		72

#define TOTAL_DATA_FILES	9
//Total number of data files or variables bird flight depends on;Does not include direction files and land water data
#define NUM_DATA_FILES		6

#define THREADS_PER_BLOCK	512
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
//------------------------------Notes---------------------------------------------------------------------------------------
/*
Altitude = 850 millibars
Year = 2009
22 Jan 2015 No upper limit to the bird flight speed currently; Birds can fly well above 10m/s
Precipitation = millimeters
*/

//--------------------------------------------------------------------------------------------------------------------------

__global__ void WrappedNormal (float* MeanAngle,float AngStdDev,float* );
__global__ void setup_kernel(unsigned int seed,hiprandState *states);
__global__ void generate_kernel(hiprandState *states,float* numbers,float* angles);
__device__ float bilinear_interpolation_SmallData(float x,float y,float* data_array);
__device__ float bilinear_interpolation_LargeData(float x,float y,float* data_array,long l);

__device__ float getProfitValue(float u_val,float v_val,float dirVal,float dir_u,float dir_v);
__device__ long int bird_AtSea(int id,int arrLength,float* rowArray,float* colArray,long l,float* udata,float* vdata,float* lwData,uint8_t* birdStatus);
__global__ void bird_movement(float* rowArray,float* colArray,int NumOfBirds,long int start_l,long int l,long int maxtimesteps,float* udata,float* vdata,float* u10data,
				float* v10data,float* dir_u,float* dir_v,float* precipData,float* pressureData,float* lwData,uint8_t* birdStatus);

static void* write_dataVars(void* arguments);
static void* read_dataFiles(void* arguments);
long int convert_to_month(int month,int day);

static void HandleError( hipError_t err,const char *file, int line );
long Get_GPU_devices();
//-------------------------------------------------------------------------------------------------------------------------------------
struct file_IO {
	FILE *fp;
	float* inpVals;
	float* streamArray;
	size_t dataSize;
}inpStruct[8]; 
//-------------------------------------------------------------------------------------------------------------------------------------
//Global Variables

float* udata;
float* vdata;
float* u10data;
float* v10data;
float* precipData;
float* pressureData;

float* dir_u;
float* dir_v;
float* lwData;

float* dirData;
//###########################################################################################################################################//

static void HandleError( hipError_t err,const char *file, int line ) {
    if (err != hipSuccess) {
  		printf( "%s in %s at line %d\n", hipGetErrorString( err ),file, line );
//		cout << hipGetErrorString(err) << "in" << file << "at line" << line << "\n";
        exit( EXIT_FAILURE );
    }
}

//###########################################################################################################################################//

long Get_GPU_devices()
{
	hipDeviceProp_t prop;
	int whichDevice,DeviceCount;
	long deviceMemory;

	HANDLE_ERROR(hipGetDevice(&whichDevice));
	HANDLE_ERROR(hipGetDeviceProperties(&prop,whichDevice));
	
	if(!prop.deviceOverlap){
			printf("Device does not handle overlaps so streams are not possible\n");
	return 0;
	}

	DeviceCount = 0;
	
	HANDLE_ERROR(hipGetDeviceCount(&DeviceCount));
	if(DeviceCount > 0){ 
		printf("%d Devices Found\n",DeviceCount);
	}else{
		printf("No devices found or error in reading the number of devices\n");
		return 0;
	}
	
	int i = 0;

	//for(int i = 0;i<DeviceCount;i++){
	hipDeviceProp_t properties;
	HANDLE_ERROR(hipGetDeviceProperties(&properties,i));
	printf("Device Number: %d\n", i);
	printf("  Device name: %s\n", properties.name);
	printf("  Device Global Memory size: %zd MB \n",properties.totalGlobalMem/1000000);
	printf("\n");
	
	deviceMemory = properties.totalGlobalMem;
	//}


	return deviceMemory;
}
//###########################################################################################################################################//

__global__ void setup_kernel(unsigned int seed,hiprandState *states)
{

	//Thread indices
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	int id = y * LONG_SIZE + x;

	hiprand_init(seed,id,0,&states[id]);
}

//###########################################################################################################################################//

__global__ void generate_kernel(hiprandState *states,float* numbers,float* angles,float* u_dirAngles,float* v_dirAngles,float speed)
{

	//Thread indices
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	int id = y * LONG_SIZE + x;
	
	float value;

	numbers[id] = hiprand_normal(&states[id]);

	if(id > (LONG_SIZE*LAT_SIZE -1)) return;
	else{
		
		u_dirAngles[id] = speed * cosf(angles[id] * (PI/180));
		v_dirAngles[id] = speed * sinf(angles[id] * (PI/180));

		value = STD_BIRDANGLE * numbers[id] + angles[id];

		if ((value - 360) > (-glCompAcc)){ 
		    value = value - 360;
		}
	 
		if (value < 0){
		    value= 360 + value;
		}
		numbers[id] = value;
		//printf("(x,y) = %d,%d,Value = %f \n",x,y,value);
	}
}

//###########################################################################################################################################//

__device__ long int bird_AtSea(int id,int arrLength,float* rowArray,float* colArray,long int l,float* udata,float* vdata,float* lwData,uint8_t* birdStatus)
{
	printf("Inside the bird_atSea() function\n");
	//long int count_timeSteps = l;
	float u_val,v_val,u_dir,v_dir,pos_row,pos_col;
	int index = 0;

	pos_row = rowArray[id * arrLength + l ];
	pos_col = colArray[id * arrLength + l ];
	
	printf("After getting the positions of row and columns\n");	
	
	//index = lwData[(int)(rintf(pos_row)) * LONG_SIZE + (int)(rintf(pos_col))];
	printf("After getting index\n");

	float count_timeSteps = 0;
	long int bckp_l;
	//int i;

	//Does not check the first time?
	//while(index != 1){
	for(count_timeSteps = 0;count_timeSteps<(BIRD_HRS_LIMIT - 10);count_timeSteps++,l++){
		
		/** Bilinear interpolation for u and v data **/
		u_val = bilinear_interpolation_LargeData(pos_col,pos_row,udata,l);	
		v_val = bilinear_interpolation_LargeData(pos_col,pos_row,vdata,l);
	
		u_dir = DESIRED_SPEED * cosf(BIRD_SEA_ANGLE * (PI/180));
		v_dir = DESIRED_SPEED * sinf(BIRD_SEA_ANGLE * (PI/180));

		/** Desired speed needs to change in the case of column position or the birds
		will not fly west **/
		pos_row = pos_row + (v_val + v_dir) * 0.36 * -1;	
		pos_col = pos_col + (u_val + u_dir) * 0.36;

		//position[(l-l_start)* PosRowLen + (id *2)] = pos_row ;
		//position[(l-l_start)* PosRowLen + (id *2) + 1] = pos_col ;
		rowArray[id * arrLength + l + 1] = pos_row;
		colArray[id * arrLength + l + 1] = pos_col;
	
		printf("Storing row and column data\n");

		index = lwData[__float2int_rd(pos_row * LAT_SIZE + pos_col)];

		if(index == 1){
			//l--;
			bckp_l = l;
			//This takes it back to the starting time of the previous day
			l = l - (6 + 4 + count_timeSteps);
			//Use casting to float to get round up value;Add to l 
			//Then do, l=l+ roundup((float)((count_timeSteps + 10)/24)) * 24; 	__float2ull_ru 
			l = l + __float2ull_ru((count_timeSteps + 10)/24) * 24 + 24 * STOPOVER_DAYS;
			
			for(bckp_l;bckp_l <= l;bckp_l++){
				rowArray[id * arrLength + bckp_l + 1 ] = pos_row;
				colArray[id * arrLength + bckp_l + 1 ] = pos_col;
			}			

			return l;
		}


		
		if(pos_row >= MAX_LAT_SOUTH){
			printf("Bird reached maximum lattitude; Exiting program\n");
			birdStatus[id] = 0;
			return -1;
		}
	}

	if(count_timeSteps >= (BIRD_HRS_LIMIT-10)){
		printf("Dead Bird! Bird has been flying for 80 hours straight!\n");
		birdStatus[id] = 0;
		return -1;
	}

	return l;
	
}

//###########################################################################################################################################//

__device__ float bilinear_interpolation_SmallData(float x,float y,float* data_array)
{
	float x1,y1,x2,y2;
	float Q11,Q12,Q21,Q22,R1,R2,R;
	//float val_x1,val_x2,val_y1,val_y2;

	x1 = floorf(x);
	x2 = ceilf(x);
	y1 = floorf(y);
	y2 = ceilf(y);
	R = 0;
	
	Q11 = data_array[(int)(y1 * LONG_SIZE + x1)];
	Q12 = data_array[(int)(y2 * LONG_SIZE + x1)];
	Q21 = data_array[(int)(y1 * LONG_SIZE + x2)];
	Q22 = data_array[(int)(y2 * LONG_SIZE + x2)];
	

	R1 = Q11 + (x - x1)*(Q21 - Q11);
	R2 = Q12 + (x - x1)*(Q22 - Q12);
	R = R1 + (y - y1)*(R2 - R1);
	
	//printf("Q11:%f,Q12:%f,Q21:%f,Q22:%f; And Value=%f\n",Q11,Q12,Q21,Q22,value);

	return R;
}


//###########################################################################################################################################//

__device__ float bilinear_interpolation_LargeData(float x,float y,float* data_array,long l)
{
	float x1,y1,x2,y2;
	float Q11,Q12,Q21,Q22,R1,R2,R;
	//float val_x1,val_x2,val_y1,val_y2;

	x1 = floorf(x);
	x2 = ceilf(x);
	y1 = floorf(y);
	y2 = ceilf(y);
	R = 0;
	
	Q11 = data_array[(int)(l  * LAT_SIZE * LONG_SIZE + y1 * LONG_SIZE + x1) ];
	Q12 = data_array[(int)(l  * LAT_SIZE * LONG_SIZE + y2 * LONG_SIZE + x1) ];
	Q21 = data_array[(int)(l  * LAT_SIZE * LONG_SIZE + y1 * LONG_SIZE + x2) ];
	Q22 = data_array[(int)(l  * LAT_SIZE * LONG_SIZE + y2 * LONG_SIZE + x2) ];
	

	R1 = Q11 + (x - x1)*(Q21 - Q11);
	R2 = Q12 + (x - x1)*(Q22 - Q12);
	R = R1 + (y - y1)*(R2 - R1);

	
	//printf("Q11:%f,Q12:%f,Q21:%f,Q22:%f; And Value=%f\n",Q11,Q12,Q21,Q22,value);
	return R;
}



//###########################################################################################################################################//

__device__ float getProfitValue(float u_val,float v_val,float dirVal,float dir_u,float dir_v)
{

	/** All wind data in m/s **/
	float diffAngle,magnitude,magnitude_squared,tailComponent,crossComponent,profit_value;

	tailComponent = 0;
	
	magnitude = hypotf(u_val,v_val);
	magnitude_squared = magnitude * magnitude;

	/** Getting the tail component of the wind; or the component of the wind in the desired direction of flight
	From formula of getting the vector projection of wind onto the desired direction **/

	tailComponent = (dir_v * v_val + dir_u * u_val);
	tailComponent = tailComponent/hypotf(dir_u,dir_u);
	

	/** DiffAngle is the angle between the desired direction of the bird and the direction of the wind
	DiffAngle has to be calculated such that both the vectors are pointing away from where they meet.
	Using the formula to get angle between two vectors **/

	diffAngle = acosf( (u_val*dir_u + v_val * dir_v)/ (( hypotf(u_val,v_val) * hypotf(dir_u,dir_v) )) ) * 180/PI;

	/** Separate profit value methods have to be used if the tail component is less that equal to or greater than the desired speed of the birds **/
	if(tailComponent <= DESIRED_SPEED) {	
		profit_value = (DESIRED_SPEED * DESIRED_SPEED) + magnitude_squared - 2 * DESIRED_SPEED * magnitude * cosf(diffAngle * PI/180);
		profit_value = DESIRED_SPEED - sqrtf(profit_value);
	}
	else {
		/** Perpendicular to a vector (x,y) is (y,-x) or (-y,x) Cross component is always positive **/

		crossComponent = fabsf((-dir_v*u_val + dir_u*v_val)/hypotf(dir_v,dir_u));
		profit_value = tailComponent - crossComponent;
	}

	return profit_value;
}

//###########################################################################################################################################//

static void* read_dataFiles(void* arguments)
{

	struct file_IO *inputArgs;
	inputArgs = (struct file_IO *)arguments;

	FILE* textFile;
	float* dataArray;


	textFile = inputArgs->fp;
	dataArray = inputArgs->inpVals;

	char line[LINESIZE];
	memset(line,'\0',sizeof(line));
	char tempVal[15];
	memset(tempVal,'\0',sizeof(tempVal));
	char* startPtr,*endPtr;
	long j;
	int i;
	float Value;
	i=0;
	j=0;

	memset(line,'\0',sizeof(line));
	memset(tempVal,'\0',sizeof(tempVal));
	i=0;
	j=0;

	while(fgets(line,LINESIZE,textFile)!=NULL){
		startPtr = line;
		for(i=0;i<LONG_SIZE;i++){
			Value = 0;
			memset(tempVal,'\0',sizeof(tempVal));

			if(i != (LONG_SIZE - 1)) {
				endPtr = strchr(startPtr,',');
				strncpy(tempVal,startPtr,endPtr-startPtr);
				//printf("%s ",tempVal);
				if(strcmp("NaN",tempVal)==0) {	
					Value = 0.0;
					
				}
				else{
					Value = atof(tempVal);
				
				}
				
				dataArray[j * LAT_SIZE + i] = Value;
				endPtr = endPtr + 1;
				startPtr = endPtr;
				//printf("%d,%f ",i,Value);
			}
			else if(i == (LONG_SIZE - 1)){
				strcpy(tempVal,startPtr);

				if(strcmp("NaN\n",tempVal)==0) {
					Value = 0.0;
				}
				else{
					Value = atof(tempVal);
				}
				dataArray[j * LAT_SIZE + i] = Value;
			}
		}
		j++;
	}
	return NULL;
}

//###########################################################################################################################################//
static void* write_dataVars(void* arguments)
{

	struct file_IO *inputArgs;
	inputArgs = (struct file_IO *)arguments;

	float* dataArray,*destArray;
	size_t totalSize;	
	long int i;

	dataArray = inputArgs->inpVals;
	destArray = inputArgs->streamArray;
	totalSize = inputArgs->dataSize;

	for(i=0;i<totalSize;i++){
		destArray[i] = *(dataArray + i);
	}

	return NULL;
}
//###########################################################################################################################################//
long int convert_to_month(int month,int day)
{
	long int index,offset;
	if(month == 8){
		index = 1; //The data starts in august
	}
	else if(month == 9){
		index = 32; //The data for september starts after 31 days of august
	}
	else if(month == 10){
		index = 62; //The data for october starts after 31+30 days of sept and august respectively.
	}
	else if(month == 11){
		index = 93; //The data for october starts after 31+30+31 days of sept,aug and oct respectively.
	}
	else{
		printf("\n\t\tIncorrect month used\n\t\tUse between August-November inclusive; Only use numbers ; August = 8\n");
		return -1;
	}

	//If 1st or 2nd of August, start at timestep 23 (after 23 hours)
	if(((month == 8) && (day == 1))||((month == 8) && (day == 2))){
		offset = 23;
	//If in August; Gives correct result for starting timestep
	}else if (month == 8){
		offset = 23 + (day - 1) * TIMESTEPS_PER_DAY ;
	//23 added because 1st day only has 23 hours
	}else{
		offset = 23 + (index - 2) * TIMESTEPS_PER_DAY + (day - 1) * TIMESTEPS_PER_DAY;
	}
	


	return offset;

}

//###########################################################################################################################################//

__global__ void bird_movement(float* rowArray,float* colArray,int NumOfBirds,long int cur_l,long int max_timesteps,float* udata,float* vdata,float* u10data,float* v10data,
				float* dir_u,float* dir_v,float* precipData,float* pressureData,float* lwData,uint8_t* birdStatus)
{

	//Thread indices
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int id = y * LONG_SIZE + x;
	//printf("Inside the kernel\n");

	if(id > (NumOfBirds -1)||(birdStatus[id]==0)||(cur_l > max_timesteps)) return;
	else{
	//Error here?
	//Make cur_l a global device variable?
	//Need to be a range with two values
		//Making a local copy of the timstep variable
		long int l;
		l = cur_l;
		printf("Value of l is %ld\n",l);

		long l_old;	
		float profit_value,actualAngle;
		float last_pressure,pressure_sum,pressure_MultSum,slope;
		float u_ten,v_ten,u_val,v_val,uDir_value,vDir_value,precip_val;
		int k,i;
		float pos_row,pos_col;
		//int current_l;
		//Length of the row and column array for each bird
		int arrLength;
		int index;

		arrLength = (TIMESTEPS + 1);
		//current_l = (int)(cur_l - l_start);
		index = id * (TIMESTEPS + 1) + l;

//		pos_row = id * arrLength + (l - l_start);
		printf("Array length per bird is %d\n",arrLength);
		printf("id is %d\n",id);
		//printf("Current l is: %d\n",current_l);
		printf("id * arrayLength is:%d\n",id*arrLength);
		printf("Calculated array index value is: %d\n",index);

		//return;
		slope = 0;

		//while(l < (TOTAL_DAYS * TIMESTEPS_PER_DAY - 24)){
		while(l < max_timesteps){
			
			//current_l = (int)(l -l_start);

			printf("Inside the while loop\n");
			//printf("Index here is %d\n",id * arrLength + current_l);
			//printf("Before printing pos_row and pos_col\n");
			printf("Starting pos_row is %f , pos_col is: %f\n",*(rowArray + id * arrLength + l),*(colArray + id * arrLength + l));
			printf("After printing pos_row and pos_col\n");
			printf("Before any computation; Timestep #: %ld\n",l);
			pos_row = rowArray[id * arrLength + l ];
			pos_col = colArray[id * arrLength + l];
		

			if((pos_row > LAT_SIZE) || (pos_col >LONG_SIZE)||(pos_row < 0)||(pos_col < 0 )){
				return;
			}
			//printf("After position calculations\n");
		
			uDir_value = dir_u[__float2int_rd(pos_row * LAT_SIZE + pos_col)];
			vDir_value = dir_v[__float2int_rd(pos_row * LAT_SIZE + pos_col)];

		
			u_ten = bilinear_interpolation_LargeData(pos_col,pos_row,u10data,l);
			v_ten = bilinear_interpolation_LargeData(pos_col,pos_row,v10data,l);

			profit_value = getProfitValue(u_ten,v_ten,actualAngle,uDir_value,vDir_value);

		
			if((profit_value >= MIN_PROFIT) && ((last_pressure>=1009)||(slope >-1))){

				//printf("Profit value greater than MIN_PROFIT\n");

				for(k=0;k<6 && l<max_timesteps;k++,l++) {
					//l = (int)(l -l_start);

					u_val = bilinear_interpolation_LargeData(pos_col,pos_row,udata,l);
					v_val = bilinear_interpolation_LargeData(pos_col,pos_row,vdata,l);
					precip_val = bilinear_interpolation_LargeData(pos_col,pos_row,precipData,l);

					//printf("End of bilinear interp for precip\n");
					//Getting new position values for row and column
					pos_row = rowArray[id * arrLength + l ];
					pos_col = colArray[id * arrLength + l ];
					//printf("Calculating row and col values\n");

					if((pos_row > LAT_SIZE) || (pos_col >LONG_SIZE)||(pos_row < 0)||(pos_col < 0 )){
						return;
					}
					//Storing the new values
					rowArray[id * arrLength + l + 1] = pos_row + (v_val + vDir_value ) * 0.36 * -1;
					colArray[id * arrLength + l + 1] = pos_col + (u_val + uDir_value) * 0.36;
			
					//printf("Storing row and col values\n");

					printf("6 Hour Flight\tRow: %f,Col:%f\n",rowArray[id * arrLength + l + 1],colArray[id * arrLength + l + 1]);
					printf("6 hour flight;Timestep #: %ld\n",l);
				}	

				printf("After 6 hour flight over\n");
				pos_row = rowArray[id * arrLength + l];
				pos_col = colArray[id * arrLength + l];
				printf("After getting row and col values\n");
				//printf("End of 6 hour flight\n");
				// If the bird is at sea after the first 6 hours of flight 
				if(lwData[__float2int_rd(pos_row * LAT_SIZE + pos_col)] != 1){

					printf("Birds at sea after 6 hours\n");
					for(k=6;k<10 && l<max_timesteps;k++,l++){
						printf("Timestep # (+4 Hours): %ld\n",l);
						// Rounding down to the nearest int 
						uDir_value = dir_u[__float2int_rd(pos_row * LAT_SIZE + pos_col)];
						vDir_value = dir_v[__float2int_rd(pos_row * LAT_SIZE + pos_col)];

						u_val = bilinear_interpolation_LargeData(pos_col,pos_row,udata,l);
						v_val = bilinear_interpolation_LargeData(pos_col,pos_row,vdata,l);
					
						//Getting new position values for row and column and storing it 
						pos_row += (v_val + vDir_value ) * 0.36 * -1;
						pos_col += (u_val + uDir_value) * 0.36;

						if((pos_row > LAT_SIZE) || (pos_col >LONG_SIZE)||(pos_row < 0)||(pos_col < 0 )){
							return;
						}
						rowArray[id * arrLength + l + 1] = pos_row;
						colArray[id * arrLength + l + 1] = pos_col;
				
						printf("+4 Hour Flight\tRow: %f,Col:%f\n",rowArray[id * arrLength + l + 1],colArray[id * arrLength + l + 1]);


					}
// If at sea even after the 4 hours 	
					if(lwData[__float2int_rd(pos_row * LAT_SIZE + pos_col)] != 1){
						printf("Birds were at sea even after 10 hours \n");
						l = bird_AtSea(id,arrLength,colArray,rowArray,l,udata,vdata,lwData,birdStatus);
						if( l == -1){
							return;
						}
						//printf("After the function bird_AtSea() \n");				
					}
					//printf("End of +4 hours of flight at sea\n");
				}else{
					for(i=6;i<24;i++,l++){		
						printf("Timestep # (Not at sea after 6 hours): %ld\n",l);			
						rowArray[id * arrLength + l + 1] = pos_row;
						colArray[id * arrLength + l + 1] = pos_col;		
					}	
				}
					
			}
			else{
				//l += 24;
				//l = (int)(l -l_start);

				for(i=0;i<18;i++,l++){		
					printf("Timestep #: %ld\n",l);			
					rowArray[id * arrLength + l + 1] = pos_row;
					colArray[id * arrLength + l + 1] = pos_col;		
				}
			}

			l_old = l - REGRESSION_HRS;

			//Taking the pressure from 6 hours earlier of the location where the bird landed
			for(k=1; (l_old < l) && (k<=REGRESSION_HRS) && (l_old<max_timesteps); l_old++,k++){

				pressure_sum += bilinear_interpolation_LargeData(pos_col,pos_row,pressureData,l_old);
				pressure_MultSum += k * bilinear_interpolation_LargeData(pos_col,pos_row,pressureData,l_old);

				//last_pressure is the last day or the day of flight
				if(k == REGRESSION_HRS) {
					last_pressure = bilinear_interpolation_LargeData(pos_col,pos_row,pressureData,l_old);
				}
			}
			slope = ((REGRESSION_HRS * pressure_MultSum) - (pressure_sum * HRS_SUM))/(DENOM_SLOPE);
		
		}		
	}
}


//###########################################################################################################################################//
int main(int argc,char* argv[])
{

//--------------------------Checking for input arguments------------------------------//

	char baseFileName[] = "../../Birds_Full/Birds_data/InterpolatedData/";
	char yearFileName[80];
	char fullFileName[80];
	char start_date[12];
	char yearStr[4],monthStr[2],dayStr[2];

	float starting_row,starting_col;
	long int offset_into_data = 0;
	int NumOfBirds,year,day,month;

	int option;
	
	while ((option = getopt(argc, argv,"y:m:d:r:c:N:")) != -1) {
        	switch (option) {
             		case 'y' : year = atoi(optarg);
             		    break;
             		case 'm' : month = atoi(optarg);
             		    break;
             		case 'd' : day = atoi(optarg); 
             		    break;
             		case 'r' : starting_row = atof(optarg);
             		    break;
             		case 'c' : starting_col = atof(optarg);
             		    break;
             //		case 't' : breadth = atoi(optarg);
             //		    break;
             		case 'N' : NumOfBirds = atoi(optarg);
             		    break;
             		default: printf("\nUsage: birds -y Year -m Month(Number) -d DayOfTheMonth -r StartingRow -c StartingCol -N NumberOfBirds\n"); 
             		    exit(EXIT_FAILURE);
        	}
   	 }

	
	/** If starting row is greater than or equal the row that we are interested in; Below a particular row we are not interested in the flight of the birds**/
	if(starting_row >= MAX_LAT_SOUTH){
		printf("\t\tProvided starting row is below the southern most lattitude at which the model is set to stop\n");
		printf("\t\tEither change the starting row location and/or MAX_LAT upto which the birds can fly\n");
		return -1;
	}
	
//-----------------------------------------------Day-----------------------------------------//
/** Making sure random date is not provided **/

	if((day>0) && (day<32)){
		sprintf(dayStr,"%d",day);
	}else{
		printf("\t\t Invalid date provided; Date should be greater than 0 and less than 32\n");
		return -1;
	}

//-----------------------------------------------Month-----------------------------------------//
/** Making sure month provided is between August and November inclusive **/

	if((month < 12) && (month > 7)){
		sprintf(monthStr,"%d",month);
	}else{
		printf("\t\t Invalid month provided; Use between 8 and 11 inclusive\n");
		return -1;
	}

	/** Converting month and day information into number of timesteps; Special case of AUG 1st is also taken care of
	Instead of AUG 1 it starts at August 2 (because data starts at 7pm but birds fly at 6pm) **/
	offset_into_data = convert_to_month(month,day);
	
	printf("Offset into data is: %ld\n",offset_into_data);

//-----------------------------------------------Year-----------------------------------------//
/** Checking if correct year specified **/

	if((year>= 2008) && (year<=2013)){
		//Add file location here
		sprintf(yearStr,"%d",year);
		strcpy(yearFileName,baseFileName);
		strcat(yearFileName,yearStr);
		strcat(yearFileName,"/");
	}
	else{
		printf("\n\tInvalid year specified\n\tSpecified %d; Use years from 2008 to 2013 in its full format\n",year);
             	printf("\t\tUsage: birds -y Year -m Month(Number) -d DayOfTheMonth -r StartingRow -c StartingCol -N NumberOfBirds\n"); 
		return -1;		
	}

	strcpy(start_date,yearStr);
	strcat(start_date,"/");	
	strcat(start_date,monthStr);
	strcat(start_date,"/");
	sprintf(dayStr,"%d",day);
	strcat(start_date,dayStr);

//------------Opening position data file where lat and long data will be stored----------------//
	
	FILE *posdataTxt,*vdataTxt,*udataTxt,*v10dataTxt,*u10dataTxt,*precipTxt,*pressureTxt,*lwTxt,*dirTxt;
	posdataTxt = fopen("posdata.txt","a");
	if(posdataTxt == NULL) {
		perror("Cannot open position data file\n");
		return -1;
	}
//----------------------Opening U850 data file----------------------------//
	memset(fullFileName,0,strlen(fullFileName));
	strcpy(fullFileName,yearFileName);
	strcat(fullFileName,"U850.txt");

	printf("U50 filename is %s \n",fullFileName);
	udataTxt = fopen(fullFileName,"r");

	if(udataTxt == NULL) {
		perror("Cannot open file with U850 data\n");
		return -1;
	}
//------------------------Opening V850 data file--------------------------//
	memset(fullFileName,0,strlen(fullFileName));
	strcpy(fullFileName,yearFileName);
	strcat(fullFileName,"V850.txt");

	vdataTxt = fopen(fullFileName,"r");

	if(vdataTxt == NULL) {
		perror("Cannot open file with V850 data\n");
		return -1;
	}
//-----------------------Opening U10 data file---------------------------//
	//Birds will check the wind at the surface therefore the u and v
	//at 10m is required
	
	memset(fullFileName,0,strlen(fullFileName));
	strcpy(fullFileName,yearFileName);
	strcat(fullFileName,"U10.txt");

	u10dataTxt = fopen(fullFileName,"r");

	if(u10dataTxt == NULL) {
		perror("Cannot open file with U10 data\n");
		return -1;
	}
//-----------------------Opening V10 data file---------------------------//
	memset(fullFileName,0,strlen(fullFileName));
	strcpy(fullFileName,yearFileName);
	strcat(fullFileName,"V10.txt");

	v10dataTxt = fopen(fullFileName,"r");
	
	if(v10dataTxt == NULL) {
		perror("Cannot open file with V10 data\n");
		return -1;
	}
//--------------------Opening PRCP data file------------------------------//
	memset(fullFileName,0,strlen(fullFileName));
	strcpy(fullFileName,yearFileName);
	strcat(fullFileName,"PRCP.txt");

	precipTxt = fopen(fullFileName,"r");
	if(precipTxt == NULL) {
		perror("Cannot open file with PRCP data\n");
		return -1;
	}
//------------------------Opening MSLP data file--------------------------//
	memset(fullFileName,0,strlen(fullFileName));
	strcpy(fullFileName,yearFileName);
	strcat(fullFileName,"MSLP.txt");

	pressureTxt = fopen(fullFileName,"r");
	if(pressureTxt == NULL) {
		perror("Cannot open file with pressure data!\n");
		return -1;
	}
//--------------------------Opening Land vs Water File---------------------//
	lwTxt = fopen("./Lw_and_Dir/land_water_detail.txt","r");
	if(lwTxt == NULL) {
		perror("Cannot open file with direction data\n");
		return -1;
	}
//--------------------------Opening Direction file 
//--------------------(Example: ext_crop.txt or extP_crop.txt)-------------//

	dirTxt = fopen("./Lw_and_Dir/ext_Final_NewCoordSystem.txt","r");
	//dirTxt = fopen("ext_crop.txt","r");
	if(dirTxt == NULL) {
		perror("Cannot open file with direction data\n");
		return -1;
	}


//-----------------------------Setting Heap Size,printf buffer size etc--------------------------------------------//
	size_t limit;
	HANDLE_ERROR(hipDeviceSetLimit(hipLimitPrintfFifoSize, 500 * 1024 * 1024));
	hipDeviceGetLimit(&limit,hipLimitPrintfFifoSize);


	HANDLE_ERROR(hipDeviceSetLimit(hipLimitMallocHeapSize,(size_t)(6 * LAT_SIZE * LONG_SIZE * TIMESTEPS * sizeof(float))));
//--------------------------Memory Allocation for global arrays containing weather data----------------------------//
	float *h_row,*h_col;
	float *d_row,*d_col;	
	float *d_udata,*d_vdata,*d_u10data,*d_v10data,*d_lwData;
	float *d_precipData,*d_pressureData;
	uint8_t *h_birdStatus,*d_birdStatus;

	dirData = (float*) malloc(LAT_SIZE * LONG_SIZE * sizeof(float));
	h_row = (float*) malloc(NumOfBirds * (TIMESTEPS + 1) * sizeof(float));
	h_col = (float*) malloc(NumOfBirds * (TIMESTEPS + 1) * sizeof(float));
	h_birdStatus = (uint8_t*)malloc(NumOfBirds * sizeof(uint8_t));

	udata = (float*)malloc(LAT_SIZE * LONG_SIZE * TIMESTEPS * sizeof(float));
	vdata = (float*)malloc(LAT_SIZE * LONG_SIZE * TIMESTEPS * sizeof(float));
	u10data = (float*)malloc(LAT_SIZE * LONG_SIZE * TIMESTEPS * sizeof(float));
	v10data = (float*)malloc(LAT_SIZE * LONG_SIZE * TIMESTEPS * sizeof(float));
	precipData = (float*)malloc(LAT_SIZE * LONG_SIZE * TIMESTEPS * sizeof(float));
	pressureData = (float*)malloc(LAT_SIZE * LONG_SIZE * TIMESTEPS * sizeof(float));
	lwData = (float*) malloc(LAT_SIZE * LONG_SIZE * sizeof(float));	

//------------------------------------------------------------------------------------------------------------------//
/*
	HANDLE_ERROR(hipHostMalloc((void**)&udata,LAT_SIZE * LONG_SIZE * TIMESTEPS * sizeof(float)));
	HANDLE_ERROR(hipHostMalloc((void**)&vdata,LAT_SIZE * LONG_SIZE * TIMESTEPS * sizeof(float)));
	HANDLE_ERROR(hipHostMalloc((void**)&u10data,LAT_SIZE * LONG_SIZE * TIMESTEPS * sizeof(float)));	
	HANDLE_ERROR(hipHostMalloc((void**)&v10data,LAT_SIZE * LONG_SIZE * TIMESTEPS * sizeof(float)));
	HANDLE_ERROR(hipHostMalloc((void**)&precipData,LAT_SIZE * LONG_SIZE * TIMESTEPS * sizeof(float)));
	HANDLE_ERROR(hipHostMalloc((void**)&pressureData,LAT_SIZE * LONG_SIZE * TIMESTEPS * sizeof(float)));
	HANDLE_ERROR(hipHostMalloc((void**)&lwData,LAT_SIZE * LONG_SIZE * sizeof(float)));	
*/

	
	printf("Size of large arrays is %zd\n",sizeof(udata)/sizeof(udata[0]));
	printf("Size of large arrays is %ld\n",sizeof(udata)/sizeof(float));
	printf("Size of large arrays is %d\n",sizeof(udata)/sizeof(float));

	int ii;
	for(ii=0;ii<(NumOfBirds * (TIMESTEPS + 1));ii++){
		*(h_row + ii) = starting_row;
		*(h_col + ii) = starting_col;
	}

	for(ii=0;ii<NumOfBirds;ii++){
		h_birdStatus[ii] = (uint8_t)1;
	}

//--------------------------Initializing the structures-------------------------------------------------------------------//

	inpStruct[0].fp = vdataTxt;
	inpStruct[0].inpVals = vdata;

	inpStruct[1].fp = udataTxt;
	inpStruct[1].inpVals = udata;

	inpStruct[2].fp = v10dataTxt;
	inpStruct[2].inpVals = v10data;

	inpStruct[3].fp = u10dataTxt;
	inpStruct[3].inpVals = u10data;

	inpStruct[4].fp = precipTxt;
	inpStruct[4].inpVals = precipData;

	inpStruct[5].fp = pressureTxt;
	inpStruct[5].inpVals = pressureData;

	inpStruct[6].fp = lwTxt;
	inpStruct[6].inpVals = lwData;

	inpStruct[7].fp = dirTxt;
	inpStruct[7].inpVals = dirData;


	/** Using pthreads to read from the files in parallel**/
	pthread_t threads[8];
	pthread_t id;

	int i,j;
	for(i=0;i<8;i++){
		if(pthread_create(&threads[i],NULL,read_dataFiles,(void*)&inpStruct[i]) != 0){
			fprintf(stderr,"ERROR: Thread creation using pthreads failed\n");
			return -1;
		}

	}

	for(i=0;i<8;i++){
		if(pthread_join(threads[i],NULL)!=0){
 			fprintf(stderr,"ERROR: Thread join failed\n");
                        return -1;
		}
	}


	printf("End of parallel data read\n");

//-----------------------------------Getting Wrapped Normal Angles-------------------------------------------//
	int DeviceCount;
	/** Getting the total number of devices available **/
	HANDLE_ERROR(hipGetDeviceCount(&DeviceCount));
	HANDLE_ERROR(hipSetDevice(DeviceCount - 1));
	HANDLE_ERROR(hipDeviceReset());

	hiprandState_t* states;
	
	HANDLE_ERROR(hipMalloc((void**)&states,LAT_SIZE*LONG_SIZE*sizeof(hiprandState_t)));

	dim3 gridSize(1,LAT_SIZE,1);
	dim3 blockSize(512,1,1);

	setup_kernel<<<gridSize,blockSize>>>(time(0),states);

	float cpu_nums[LAT_SIZE * LONG_SIZE];
	float *rand_norm_nums,*d_dirData,*d_u_dirAngle,*d_v_dirAngle;

	HANDLE_ERROR(hipMalloc((void**)&rand_norm_nums,LAT_SIZE*LONG_SIZE*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&d_dirData,LAT_SIZE*LONG_SIZE*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&d_u_dirAngle,LAT_SIZE*LONG_SIZE*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&d_v_dirAngle,LAT_SIZE*LONG_SIZE*sizeof(float)));

	//Trying to access pinned memory here; Needs more args in the kernel?
	//Memcpy async?; Or just malloc dirData instead of making it pinned
	HANDLE_ERROR(hipMemcpy(d_dirData,dirData, LAT_SIZE * LONG_SIZE * sizeof(float), hipMemcpyHostToDevice));
	generate_kernel<<<gridSize,blockSize>>>(states,rand_norm_nums,d_dirData,d_u_dirAngle,
						d_v_dirAngle,(float)DESIRED_SPEED);

//Do not need to get them back at all; Will have to send it back to GPU 
//	hipMemcpy(cpu_nums,rand_norm_nums, LAT_SIZE * LONG_SIZE * sizeof(float),hipMemcpyDeviceToHost);
//	hipMemcpy(dir_u,d_u_dirAngle,LAT_SIZE * LONG_SIZE * sizeof(float),hipMemcpyDeviceToHost);
//	hipMemcpy(dir_v,d_v_dirAngle,LAT_SIZE * LONG_SIZE * sizeof(float),hipMemcpyDeviceToHost);

	/* print them out */
/*	for ( j = 0; j < LAT_SIZE; j++) {
		for( i = 0;i<LONG_SIZE;i++){
			//printf("%f ", cpu_nums[j*LONG_SIZE + i]);
			if(i == LONG_SIZE -1) {
				printf("%f\n",dir_u[j * LAT_SIZE + i]);
			}
			else {
				printf("%f ",dir_u[j * LAT_SIZE + i]);
			}
		}
//		printf("\n");
	}
*/

	HANDLE_ERROR(hipDeviceSynchronize());

	// free the memory we allocated for the states and numbers 
	HANDLE_ERROR(hipFree(states));
	HANDLE_ERROR(hipFree(rand_norm_nums));
	HANDLE_ERROR(hipFree(d_dirData));
	
	free(dirData);


	printf("Random number generator is working\n");

//-------------------------------------------------------------------------------------------------------------------------//	
	HANDLE_ERROR(hipMalloc((void**)&d_row,NumOfBirds * (TIMESTEPS + 1 ) * sizeof(float)));	
	HANDLE_ERROR(hipMalloc((void**)&d_col,NumOfBirds * (TIMESTEPS + 1 ) * sizeof(float)));	
	HANDLE_ERROR(hipMalloc((void**)&d_lwData,LAT_SIZE * LONG_SIZE * sizeof(float)));
	

	HANDLE_ERROR(hipMemcpy(d_row,h_row,NumOfBirds * (TIMESTEPS + 1 ) * sizeof(float),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_col,h_col,NumOfBirds * (TIMESTEPS + 1 ) * sizeof(float),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_lwData,lwData,LAT_SIZE * LONG_SIZE * sizeof(float),hipMemcpyHostToDevice));
	
//-------------------------------------------------------------------------------------------------------------//	
	size_t MemoryEachVar,DataPerTransfer,SizePerTimestep;
	int TimestepsPerTransfer,DaysPerTransfer;		
	size_t MemoryRemaining,TotalMemory;

	HANDLE_ERROR(hipSetDevice(DeviceCount - 1));

	// Getting the total remaining memory that the device can allocate 
	HANDLE_ERROR(hipMemGetInfo(&MemoryRemaining,&TotalMemory));

	MemoryRemaining -= 2*NumOfBirds* (TIMESTEPS + 1) * sizeof(float);
	MemoryRemaining -= NumOfBirds * sizeof(uint8_t);	
	//Need to make sure 100MB is free!! For some reason
	MemoryRemaining -= 100 * 1000000;

	
	printf("Total mem: %zd,Free mem: %zd\n",TotalMemory,MemoryRemaining);

 	printf("\n\n\t\t Total Memory remaining is: %zd \n",MemoryRemaining);

	//Memory that each variable gets every timestep
	MemoryEachVar = MemoryRemaining/NUM_DATA_FILES;

	printf("\t\t Memory for each variable is: %zd \n",MemoryEachVar);

	// Need to send data per timestep so has to be a multiple of LAT_SIZE *LONG_SIZE* sizeof(float) * 24
	//Can also be called as Minimum_Size_Per_Timestep; Sending data so that it is according to days
	SizePerTimestep = LAT_SIZE * LONG_SIZE * TIMESTEPS_PER_DAY * sizeof(float);

	// To get a number divisible by SizePerTimestep
	//DataPerTransfer is the data size to be transferred for each variable
	//Example, if 100MB then 100MB for each of the vars is transferred each time
	DataPerTransfer = (MemoryEachVar/SizePerTimestep) * SizePerTimestep;
	DaysPerTransfer = DataPerTransfer/SizePerTimestep;
	TimestepsPerTransfer = DaysPerTransfer * TIMESTEPS_PER_DAY;

	printf("\t\tChecking Division: %zd\n",MemoryEachVar/SizePerTimestep);		
	printf("\t\t Total Timesteps per Transfer of data is: %ld \n",TimestepsPerTransfer); 
	printf("\t\tData per transfer is %zd\n",DataPerTransfer);	
	
//------------------------------------Getting the size of data needed per transfer---------------------------------------------//
	int divisible,Transfers;
	long int DataLastTransfer;//Per variable

	Transfers = (TOTAL_DAYS * TIMESTEPS_PER_DAY) / TimestepsPerTransfer;

	divisible = (TOTAL_DAYS*TIMESTEPS_PER_DAY) % TimestepsPerTransfer;
	
	if(divisible != 0){
			Transfers++;
	}
	
	printf("Total Transfers required: %ld\n",Transfers);
	/** Tota bytes transfered per data transfer**/

	const int TotalTransfers = Transfers;
/*
	hipStream_t stream[TotalTransfers-1];
	for(i=0;i<TotalTransfers-1;i++){
		HANDLE_ERROR(hipStreamCreate(&stream[i]));
	}
*/
	DataLastTransfer = TOTAL_DAYS * TIMESTEPS_PER_DAY * LAT_SIZE * LONG_SIZE * sizeof(float) 
								- DataPerTransfer * (TotalTransfers-1); 

//---------------------------------------Memory allocation per transfer----------------------------------------------------------//
	
	long int ptrOffset;
	//ptrOffset = INITIAL_SKIP_TIMESTEPS;	
	ptrOffset = 0;


	long int cur_timestep,max_timesteps;

	//min_timesteps = offset_into_data;
	//printf("Current timestep variable is:%ld\n",min_timesteps);
	//return 0;
	cur_timestep = offset_into_data;

	//printf("cur_timestep = offset_into_data; Value in cur_timestep is: %ld\n",cur_timestep);

	for(i=0;i<TotalTransfers-1;i++){


		HANDLE_ERROR(hipSetDevice(DeviceCount - 1));
		HANDLE_ERROR(hipMemGetInfo(&MemoryRemaining,&TotalMemory));
		printf("Total mem: %zd,Free mem(Before any allocation): %zd\n",TotalMemory,MemoryRemaining);

		
		HANDLE_ERROR(hipMemGetInfo(&MemoryRemaining,&TotalMemory));
		printf("Total mem: %zd,Free mem(After SetDevice): %zd\n",TotalMemory,MemoryRemaining);

		//HANDLE_ERROR(hipStreamCreate(&stream[i]));
		HANDLE_ERROR(hipMemGetInfo(&MemoryRemaining,&TotalMemory));
		printf("Total mem: %zd,Free mem(After Stream Create): %zd\n",TotalMemory,MemoryRemaining);

		HANDLE_ERROR(hipMalloc((void**)&d_udata,DataPerTransfer));	
		HANDLE_ERROR(hipMemGetInfo(&MemoryRemaining,&TotalMemory));
		printf("Total mem: %zd,Free mem(After udata allocation): %zd\n",TotalMemory,MemoryRemaining);
	
		HANDLE_ERROR(hipMalloc((void**)&d_vdata,DataPerTransfer));	
		HANDLE_ERROR(hipMemGetInfo(&MemoryRemaining,&TotalMemory));
		printf("Total mem: %zd,Free mem(After vdata allocation): %zd\n",TotalMemory,MemoryRemaining);

		HANDLE_ERROR(hipMalloc((void**)&d_u10data,DataPerTransfer));	
		HANDLE_ERROR(hipMemGetInfo(&MemoryRemaining,&TotalMemory));
		printf("Total mem: %zd,Free mem(After u10data allocation): %zd\n",TotalMemory,MemoryRemaining);

		HANDLE_ERROR(hipMalloc((void**)&d_v10data,DataPerTransfer));	
		HANDLE_ERROR(hipMemGetInfo(&MemoryRemaining,&TotalMemory));
		printf("Total mem: %zd,Free mem(After v10data allocation): %zd\n",TotalMemory,MemoryRemaining);

		HANDLE_ERROR(hipMalloc((void**)&d_precipData,DataPerTransfer));	
		HANDLE_ERROR(hipMemGetInfo(&MemoryRemaining,&TotalMemory));
		printf("Total mem: %zd,Free mem(After precipData allocation): %zd\n",TotalMemory,MemoryRemaining);

		HANDLE_ERROR(hipMalloc((void**)&d_pressureData,DataPerTransfer));	
		HANDLE_ERROR(hipMemGetInfo(&MemoryRemaining,&TotalMemory));
		printf("Total mem: %zd,Free mem(After pressureData allocation): %zd\n",TotalMemory,MemoryRemaining);
	
		HANDLE_ERROR(hipMalloc((void**)&d_birdStatus,NumOfBirds * sizeof(uint8_t)));
		HANDLE_ERROR(hipMemGetInfo(&MemoryRemaining,&TotalMemory));
		printf("Total mem: %zd,Free mem(After pressureData allocation): %zd\n",TotalMemory,MemoryRemaining);
	
		HANDLE_ERROR(hipDeviceSynchronize());


		printf("After all the host allocations %d\n",i);



	//-----------------------------------------Initializing gridSize and block Size-------------------------------//		
		//HANDLE_ERROR(hipSetDevice(DeviceCount - 1));

		dim3 gridSize((NumOfBirds + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK,1,1);
		dim3 blockSize(THREADS_PER_BLOCK,1,1);

		HANDLE_ERROR(hipMemGetInfo(&MemoryRemaining,&TotalMemory));
		printf("Total mem: %zd,Free mem(After grid and block init): %zd\n",TotalMemory,MemoryRemaining);
	//-----------------------------------------Copying data from CPU to GPU------------------------------------------------//	

		HANDLE_ERROR(hipSetDevice(DeviceCount - 1));	

		HANDLE_ERROR(hipMemcpy(d_udata,udata+ptrOffset,DataPerTransfer,hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(d_vdata,vdata+ptrOffset,DataPerTransfer,hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(d_u10data,u10data+ptrOffset,DataPerTransfer,hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(d_v10data,v10data+ptrOffset,DataPerTransfer,hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(d_precipData,precipData+ptrOffset,DataPerTransfer,hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(d_pressureData,pressureData+ptrOffset,DataPerTransfer,hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(d_birdStatus,h_birdStatus,NumOfBirds * sizeof(uint8_t),hipMemcpyHostToDevice));

/*
		HANDLE_ERROR(hipMemcpyAsync(d_lwData,lwData,LAT_SIZE * LONG_SIZE * sizeof(float),hipMemcpyHostToDevice,stream[i]));
		HANDLE_ERROR(hipMemGetInfo(&MemoryRemaining,&TotalMemory));
		printf("Total mem: %zd,Free mem(After grid and block init): %zd\n",TotalMemory,MemoryRemaining);
		HANDLE_ERROR(hipMemcpyAsync(d_udata,udata + ptrOffset,DataPerTransfer,hipMemcpyHostToDevice,stream[i]));
		HANDLE_ERROR(hipMemcpyAsync(d_vdata,(vdata+ptrOffset),DataPerTransfer,hipMemcpyHostToDevice,stream[i]));
		HANDLE_ERROR(hipMemcpyAsync(d_u10data,(u10data+ptrOffset),DataPerTransfer,hipMemcpyHostToDevice,stream[i]));
		HANDLE_ERROR(hipMemcpyAsync(d_v10data,(v10data+ptrOffset),DataPerTransfer,hipMemcpyHostToDevice,stream[i]));
		HANDLE_ERROR(hipMemcpyAsync(d_precipData,(precipData+ptrOffset),DataPerTransfer,hipMemcpyHostToDevice,stream[i]));
		HANDLE_ERROR(hipMemcpyAsync(d_pressureData,(pressureData+ptrOffset),DataPerTransfer,hipMemcpyHostToDevice,stream[i]));
*/
	//-----------------------------------------Calling the Kernel-----------------------------------------------------------//
		
		//All of these are inclusive
		//If TimeStepsPerTransfer is 9, then they would be: 0-8, 9-17, 18-26,...
		max_timesteps = ((i+1) * TimestepsPerTransfer) - 1;
		

		printf("Current timestep variable is:%ld\n",cur_timestep);
		printf("Max timestep is: %ld\n",max_timesteps);
		printf("Offset into data is:%ld\n",offset_into_data);


		if((offset_into_data <= max_timesteps) && (i > 0)){
			offset_into_data = i * TimestepsPerTransfer;
			cur_timestep = offset_into_data;
		}
		printf("Current timestep variable after checking if offset less than max_timesteps is:%ld\n",cur_timestep);

		bird_movement<<<gridSize,blockSize>>>(d_row,d_col,NumOfBirds,cur_timestep,max_timesteps,d_udata,d_vdata,
						d_u10data,d_v10data,d_u_dirAngle,d_v_dirAngle,d_precipData,d_pressureData,d_lwData,d_birdStatus);


		//HANDLE_ERROR(hipStreamSynchronize(stream[i]));
		HANDLE_ERROR(hipDeviceSynchronize());
	//---------------------------------Freeing allocated memory in GPU and pinned memory in CPU-------------------//
		printf("Before freeing;Inside the loop\n");

		HANDLE_ERROR(hipMemcpy(h_birdStatus,d_birdStatus,NumOfBirds * sizeof(uint8_t),hipMemcpyDeviceToHost));



		//HANDLE_ERROR(hipStreamDestroy(stream[i]));
//		HANDLE_ERROR(hipFree(d_lwData));
		//HANDLE_ERROR(hipFree(d_birdStatus));		
		HANDLE_ERROR(hipFree(d_udata));
		HANDLE_ERROR(hipFree(d_vdata));
		HANDLE_ERROR(hipFree(d_u10data));
		HANDLE_ERROR(hipFree(d_v10data));
		HANDLE_ERROR(hipFree(d_precipData));
		HANDLE_ERROR(hipFree(d_pressureData));

		
		//ptrOffset+= DataPerTransfer/sizeof(float); 
		ptrOffset = (DataPerTransfer/sizeof(float)) * (i + 1);
		printf("After all freeing %d\n",i);
		
	}
/*
	HANDLE_ERROR(hipMemcpy(h_row,d_row,NumOfBirds * (TIMESTEPS + 1 ) * sizeof(float),hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(h_col,d_col,NumOfBirds * (TIMESTEPS + 1 ) * sizeof(float),hipMemcpyDeviceToHost));
	

	for(i = 0;i < NumOfBirds * (TIMESTEPS + 1); i++ ){
		printf("%f ",h_row[i]);	
		if(i == TIMESTEPS){
			printf("%f \n",h_row[i]);
		}

	}


	printf("\n\n");
	for(i = 0;i < NumOfBirds * (TIMESTEPS + 1); i++ ){
		printf("%f ",h_col[i]);	
		if(i == TIMESTEPS){
			printf("%f \n",h_col[i]);
		}

	}
*/
//----------------------------------------------------Last Iteration-----------------------------------------//


	// Last iteration where the size might not be the same as others 
	long int DataRemaining;
	DataRemaining = LONG_SIZE * LAT_SIZE * TIMESTEPS * sizeof(float) - (DataPerTransfer * (TotalTransfers-1));
 	DataRemaining = DataRemaining/NUM_DATA_FILES;


	max_timesteps = TIMESTEPS;
	cur_timestep = (TotalTransfers - 1) * TimestepsPerTransfer;
	ptrOffset = (DataPerTransfer/sizeof(float)) * (TotalTransfers - 1);
 
	HANDLE_ERROR(hipSetDevice(DeviceCount - 1));
	HANDLE_ERROR(hipMemGetInfo(&MemoryRemaining,&TotalMemory));
	printf("Total mem: %zd,Free mem(Before any allocation): %zd\n",TotalMemory,MemoryRemaining);

	
	HANDLE_ERROR(hipMemGetInfo(&MemoryRemaining,&TotalMemory));
	printf("Total mem: %zd,Free mem(After SetDevice): %zd\n",TotalMemory,MemoryRemaining);

	//HANDLE_ERROR(hipStreamCreate(&stream[i]));
	HANDLE_ERROR(hipMemGetInfo(&MemoryRemaining,&TotalMemory));
	printf("Total mem: %zd,Free mem(After Stream Create): %zd\n",TotalMemory,MemoryRemaining);

	HANDLE_ERROR(hipMalloc((void**)&d_udata,DataRemaining));	
	HANDLE_ERROR(hipMemGetInfo(&MemoryRemaining,&TotalMemory));
	printf("Total mem: %zd,Free mem(After udata allocation): %zd\n",TotalMemory,MemoryRemaining);

	HANDLE_ERROR(hipMalloc((void**)&d_vdata,DataRemaining));	
	HANDLE_ERROR(hipMemGetInfo(&MemoryRemaining,&TotalMemory));
	printf("Total mem: %zd,Free mem(After vdata allocation): %zd\n",TotalMemory,MemoryRemaining);

	HANDLE_ERROR(hipMalloc((void**)&d_u10data,DataRemaining));	
	HANDLE_ERROR(hipMemGetInfo(&MemoryRemaining,&TotalMemory));
	printf("Total mem: %zd,Free mem(After u10data allocation): %zd\n",TotalMemory,MemoryRemaining);

	HANDLE_ERROR(hipMalloc((void**)&d_v10data,DataRemaining));	
	HANDLE_ERROR(hipMemGetInfo(&MemoryRemaining,&TotalMemory));
	printf("Total mem: %zd,Free mem(After v10data allocation): %zd\n",TotalMemory,MemoryRemaining);

	HANDLE_ERROR(hipMalloc((void**)&d_precipData,DataRemaining));	
	HANDLE_ERROR(hipMemGetInfo(&MemoryRemaining,&TotalMemory));
	printf("Total mem: %zd,Free mem(After precipData allocation): %zd\n",TotalMemory,MemoryRemaining);

	HANDLE_ERROR(hipMalloc((void**)&d_pressureData,DataRemaining));	
	HANDLE_ERROR(hipMemGetInfo(&MemoryRemaining,&TotalMemory));
	printf("Total mem: %zd,Free mem(After pressureData allocation): %zd\n",TotalMemory,MemoryRemaining);

	//HANDLE_ERROR(hipMalloc((void**)&d_birdStatus,NumOfBirds * sizeof(uint8_t)));
	//HANDLE_ERROR(hipMemGetInfo(&MemoryRemaining,&TotalMemory));
	//printf("Total mem: %zd,Free mem(After pressureData allocation): %zd\n",TotalMemory,MemoryRemaining);

	HANDLE_ERROR(hipDeviceSynchronize());


	printf("After all the host allocations %d\n",i);
//-----------------------------------------Initializing gridSize and block Size-------------------------------//	

	printf("Before grid and block size allocations\n");

	//dim3 gridSize2((NumOfBirds + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK,1,1);
	//dim3 blockSize2(THREADS_PER_BLOCK,1,1);

	printf("After grid and block size allocations\n");

//-----------------------------------------Copying data from CPU to GPU----------------------------------------//

	HANDLE_ERROR(hipSetDevice(DeviceCount - 1));	

	HANDLE_ERROR(hipMemcpy(d_udata,udata+ptrOffset,DataRemaining,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_vdata,vdata+ptrOffset,DataRemaining,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_u10data,u10data+ptrOffset,DataRemaining,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_v10data,v10data+ptrOffset,DataRemaining,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_precipData,precipData+ptrOffset,DataRemaining,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_pressureData,pressureData+ptrOffset,DataRemaining,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_birdStatus,h_birdStatus,NumOfBirds * sizeof(uint8_t),hipMemcpyHostToDevice));

//-----------------------------------------Calling the Kernel-------------------------------------------------//

	printf("Before calling the kernel\n");
	bird_movement<<<gridSize,blockSize>>>(d_row,d_col,NumOfBirds,cur_timestep,max_timesteps,d_udata,d_vdata,
						d_u10data,d_v10data,d_u_dirAngle,d_v_dirAngle,d_precipData,d_pressureData,d_lwData,d_birdStatus);

	HANDLE_ERROR(hipDeviceSynchronize());



	HANDLE_ERROR(hipMemcpy(h_row,d_row,NumOfBirds * (TIMESTEPS + 1 ) * sizeof(float),hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(h_col,d_col,NumOfBirds * (TIMESTEPS + 1 ) * sizeof(float),hipMemcpyDeviceToHost));
	

	for(i = 0;i < NumOfBirds * (TIMESTEPS + 1); i++ ){
		printf("%f ",h_row[i]);	
		if(((i+1) % (TIMESTEPS + 1)) == 0){
			printf("%f \n",h_row[i]);
		}

	}


	printf("\n\n");
	for(i = 0;i < NumOfBirds * (TIMESTEPS + 1); i++ ){
		printf("%f ",h_col[i]);	
		if(((i+1) % (TIMESTEPS + 1)) == 0){
			printf("%f \n",h_col[i]);
		}

	}
//-----------------------------------------------Freeing allocated memory--------------------------------------//
//	HANDLE_ERROR(hipStreamDestroy(stream[0]));
	HANDLE_ERROR(hipFree(d_birdStatus));		
	HANDLE_ERROR(hipFree(d_udata));
	HANDLE_ERROR(hipFree(d_vdata));
	HANDLE_ERROR(hipFree(d_u10data));
	HANDLE_ERROR(hipFree(d_v10data));
	HANDLE_ERROR(hipFree(d_precipData));
	HANDLE_ERROR(hipFree(d_pressureData));
/*	
	HANDLE_ERROR(hipHostFree(udata));
	HANDLE_ERROR(hipHostFree(vdata));
	HANDLE_ERROR(hipHostFree(u10data));
	HANDLE_ERROR(hipHostFree(v10data));
	HANDLE_ERROR(hipHostFree(precipData));
	HANDLE_ERROR(hipHostFree(pressureData));
	HANDLE_ERROR(hipHostFree(lwData));
*/


	free(udata);
	free(vdata);
	free(u10data);
	free(v10data);
	free(precipData);
	free(pressureData);
	free(lwData);
	free(h_birdStatus);
/*
	HANDLE_ERROR(hipFree(d_lwData));	
	HANDLE_ERROR(hipFree(d_u_dirAngle));
	HANDLE_ERROR(hipFree(d_v_dirAngle));
	printf("After freeing everything\n");
*/
	HANDLE_ERROR(hipFree(d_row));	
	HANDLE_ERROR(hipFree(d_col));	
	free(h_row);
	free(h_col);
	//free(lwData);
	//free(dirData);
	
	fclose(dirTxt);
	fclose(posdataTxt);	
	fclose(udataTxt);
	fclose(vdataTxt);
	fclose(v10dataTxt);
	fclose(u10dataTxt);
	fclose(precipTxt);
	fclose(pressureTxt);
	fclose(lwTxt);
	
	printf("End\n");
	return 0;
}
