#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define PI 3.14159

#define LONG_SIZE		95
#define LAT_SIZE		55
#define LINESIZE	8*LAT_SIZE+LAT_SIZE - 3

#define TIMESTEPS	720
#define SKIP_TIMESTEPS	23
//#define DESIRED_ROW	
//#define DESIRED_COL
#define STARTING_ROW	11
#define STARTING_COL	14

#define DESIREDANGLE	90
#define DESIRED_SPEED	36	//In km/hr



//Altitude = 850 millibars
//Year = 1980

//--------------------------------------------------------------------------------------------------------------------------

__global__ void get_resultant(float * u, float* v,float* resultantMatrix,float* resultantAngle);
void get_movementData(FILE* outTxt,float* udata,float* vdata);
float getAngleValue(float u,float v);

//--------------------------------------------------------------------------------------------------------------------------

//Kernel to get angle and magnitude from u and v matrices
__global__ void get_resultant(float * u, float* v,float* resultantMatrix,float* resultantAngle)
{
	float angle;

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + y * LAT_SIZE; 

	if(x < LONG_SIZE) {
		resultantMatrix[index] = hypotf(u[index],v[index]);
		if((v[index] >0)&&( u[index] >0)) {
			angle = atanf(u[index]/v[index]) * (180/PI);
		}
		else if ((v[index] > 0)&&( u[index] < 0)){
			angle = 360 - atanf(u[index]/v[index]) * (180/PI);
		}
		else if ((v[index] < 0)&&( u[index] > 0)){
			angle = 180 - atanf(u[index]/v[index]) * (180/PI);
		}
		else if ((v[index] < 0)&&( u[index] < 0)){
			angle = 180 + atanf(u[index]/v[index]) * (180/PI);
		}
		else if ((v[index] == 0)&&( u[index] > 0)){
			angle = 90;
		}
		else if ((v[index] == 0)&&( u[index] < 0)){
			angle = 270;
		}
		else if ((v[index] > 0)&&( u[index] == 0)){
			angle = 0;
		}
		else if ((v[index] < 0)&&( u[index] == 0)){
			angle = 180;
		}
		if(angle < 0){
			angle = (360 - angle);
		}
		if(angle > 360) {
			angle = angle - 360;
		}
		resultantAngle[index] = angle;
		//printf("%f,%f\n",resultantMatrix[index],angle);
	}	

}

/*
//Computation Kernel
__global__ void bird_thread(float* resultantMatrix,float* resultantAngle,int* coords_row,int* coords_col)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;	

	int pos_row,pos_col;
	pos_row = 90;
	pos_col = 25;
	
	float tempAngle = 0;
	int i;

	for(i = 0;i<TIMESTEPS;i++) {
		tempAngle = resultantAngle[pos_row * LONG_SIZE + pos_col];
		if((tempAngle >= 0) && (tempAngle < 45)) {
			pos_row -= 1;
		} 
		else if((tempAngle >= 45) && (tempAngle < 90)) {
			pos_row -= 1;
			pos_col += 1;
		}
		else if((tempAngle >= 90) && (tempAngle < 135)) {
			pos_col += 1;
		}
		else if((tempAngle >= 135) && (tempAngle < 180)) {
			pos_row += 1;
			pos_col += 1;
		}
		else if((tempAngle >= 180) && (tempAngle < 225)) {
			pos_row += 1;
		}
		else if((tempAngle >= 225) && (tempAngle < 270)) {
			pos_row += 1;
			pos_col -= 1;
		}
		else if((tempAngle >= 270) && (tempAngle < 315)) {
			pos_col -= 1;
		}
		else if((tempAngle >= 315) && (tempAngle < 360)) {
			pos_row -= 1;
			pos_col -= 1;
		}
		
		coords_row[TIMESTEPS] = pos_row;
		coords_col[TIMESTEPS] = pos_col;
		printf("%d,%d\n",pos_row,pos_col);		
	
	}
}
*/


//--------------------------------------------------------------------------------------------------------------------------
int main()
{

	size_t limit;
	hipDeviceSetLimit(hipLimitPrintfFifoSize, 500 * 1024 * 1024);
	hipDeviceSetLimit(hipLimitMallocHeapSize, 128*1024*1024);
	hipDeviceGetLimit(&limit,hipLimitPrintfFifoSize);

	float* udata;
	udata = (float*)malloc(LAT_SIZE  * LONG_SIZE * TIMESTEPS * sizeof(float));
	float* vdata;
	vdata = (float*)malloc(LAT_SIZE  * LONG_SIZE * TIMESTEPS * sizeof(float));
	float* dirData;
	dirData = (float*)malloc(LAT_SIZE  * LONG_SIZE * sizeof(float));
	

	FILE *posdataTxt;
	posdataTxt = fopen("posdata.txt","w");
	if(posdataTxt == NULL) {
		perror("Cannot open udataTxt file\n");
		return -1;
	}


	FILE *vdataTxt,*udataTxt;
	udataTxt = fopen("uvalue.txt","r");
	vdataTxt = fopen("vvalue.txt","r");
	if(udataTxt == NULL) {
		perror("Cannot open udataTxt file\n");
		return -1;
	}
	if(vdataTxt == NULL) {
		perror("Cannot open udataTxt file\n");
		return -1;
	}

	FILE* dirTxt;
	dirTxt = fopen("direction.txt","r");
	if(dirTxt == NULL) {
		perror("Cannot open dirTxt file\n");
		return -1;
	}

	FILE* inpCheckU;
	inpCheckU = fopen("inpCheckU.txt","w");
	if(inpCheckU == NULL) {
		perror("Cannot open udataTxt file\n");
		return -1;
	}
	
	FILE* inpCheckV;
	inpCheckV = fopen("inpCheckV.txt","w");
	if(inpCheckV == NULL) {
		perror("Cannot open udataTxt file\n");
		return -1;
	}
	
	char line[LINESIZE];
	memset(line,'\0',sizeof(line));

	char tempVal[8];
	memset(tempVal,'\0',sizeof(tempVal));

	char* startPtr,*endPtr;

	long j;
	int i;
	float Value;
	
	i=0;
	j=0;
	
	while(fgets(line,LINESIZE,udataTxt)!=NULL){
		startPtr = line;
		for(i=0;i<LAT_SIZE;i++){
			Value = 0;
			memset(tempVal,'\0',sizeof(tempVal));

			if(i != (LAT_SIZE - 1)) {
				endPtr = strchr(startPtr,',');
				strncpy(tempVal,startPtr,endPtr-startPtr);
				Value = atof(tempVal);
				udata[j * LAT_SIZE + i] = Value;
				endPtr = endPtr + 1;
				startPtr = endPtr;
				
			}
			else if(i == (LAT_SIZE - 1)){

				strcpy(tempVal,startPtr);
				Value = atof(tempVal);
				udata[j * LAT_SIZE + i] = Value;
				
			}	
					
		}
		j++;
	}	

	

	memset(line,'\0',sizeof(line));
	memset(tempVal,'\0',sizeof(tempVal));
	
	i=0;
	j=0;
	
	while(fgets(line,LINESIZE,vdataTxt)!=NULL){
		startPtr = line;
		for(i=0;i<LAT_SIZE;i++){
			Value = 0;
			memset(tempVal,'\0',sizeof(tempVal));

			if(i != (LAT_SIZE - 1)) {

				endPtr = strchr(startPtr,',');
				strncpy(tempVal,startPtr,endPtr-startPtr);
				Value = atof(tempVal);
				vdata[j * LAT_SIZE + i] = Value;
				endPtr = endPtr + 1;
				startPtr = endPtr;
				
			}
			else if(i == (LAT_SIZE - 1)){

				strcpy(tempVal,startPtr);
				Value = atof(tempVal);
				vdata[j * LAT_SIZE + i] = Value;
				
			}			
		}
		j++;
	}
	

	memset(line,'\0',sizeof(line));
	memset(tempVal,'\0',sizeof(tempVal));
	
	i=0;
	j=0;

	while(fgets(line,LINESIZE,dirTxt)!=NULL){
		startPtr = line;
		for(i=0;i<LAT_SIZE;i++){
			Value = 0;
			memset(tempVal,'\0',sizeof(tempVal));

			if(i != (LAT_SIZE - 1)) {

				endPtr = strchr(startPtr,',');
				strncpy(tempVal,startPtr,endPtr-startPtr);
				Value = atof(tempVal);
				dirData[j * LAT_SIZE + i] = Value;
				endPtr = endPtr + 1;
				startPtr = endPtr;
			
			}
			else if(i == (LAT_SIZE - 1)){

				strcpy(tempVal,startPtr);
				Value = atof(tempVal);
				dirData[j * LAT_SIZE + i] = Value;
			
			}	
			printf("%f\n",dirData[j * LAT_SIZE + i]);		
		}
		j++;
	}

	for(j=0;j<LONG_SIZE * TIMESTEPS;j++) {
		for(i=0;i<LAT_SIZE;i++) {
			if(i == LAT_SIZE -1) {
				fprintf(inpCheckU,"%f\n",udata[j * LAT_SIZE + i]);
				fprintf(inpCheckV,"%f\n",vdata[j * LAT_SIZE + i]);
			}
			else {
				fprintf(inpCheckU,"%f ",udata[j * LAT_SIZE + i]);
				fprintf(inpCheckV,"%f ",vdata[j * LAT_SIZE + i]);
			}
		}
	}
	get_movementData(posdataTxt,udata,vdata);
/*
	float resultantMatrix[LONG_SIZE * LAT_SIZE];
	float resultantAngle[LONG_SIZE * LAT_SIZE];
	
	resultantMatrix[LAT_SIZE * LONG_SIZE -1] = 834.0;	

	float *udataPtr,*vdataPtr,*resultantMatrixPtr,*resultantAnglePtr;

	hipMalloc((void**)&udataPtr,LONG_SIZE * LAT_SIZE * sizeof(float));
	hipMemcpy(udataPtr,udata,LONG_SIZE * LAT_SIZE * sizeof(float),hipMemcpyHostToDevice);

	hipMalloc((void**)&vdataPtr,LONG_SIZE * LAT_SIZE * sizeof(float));
	hipMemcpy(vdataPtr,vdata,LONG_SIZE * LAT_SIZE * sizeof(float),hipMemcpyHostToDevice);

	hipMalloc((void**)&resultantMatrixPtr,LONG_SIZE * LAT_SIZE * sizeof(float));
	hipMalloc((void**)&resultantAnglePtr,LONG_SIZE * LAT_SIZE * sizeof(float));
	
	hipMemcpy(vdataPtr,vdata,LONG_SIZE * LAT_SIZE * sizeof(float),hipMemcpyHostToDevice);

	dim3 gridSize(1,LONG_SIZE,1);
	dim3 blockSize((LAT_SIZE/32 +1)*32 ,1,1);


	printf("Hello2\n");
	get_resultant<<<gridSize,blockSize>>>(udataPtr,vdataPtr,resultantMatrixPtr,resultantAnglePtr);
	printf("Hello3\n");
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
  	{
		printf("CUDA Error: %s\n", hipGetErrorString(error));

    	// we can't recover from the error -- exit the program
    	return 0;
  	}

	hipMemcpy(resultantMatrix,resultantMatrixPtr,LAT_SIZE * LONG_SIZE * sizeof(float),hipMemcpyDeviceToHost);
	hipMemcpy(resultantAngle,resultantAnglePtr,LAT_SIZE * LONG_SIZE * sizeof(float),hipMemcpyDeviceToHost);

	hipFree(udataPtr);
	hipFree(vdataPtr);
	hipFree(resultantMatrixPtr);
	hipFree(resultantAnglePtr);
	
	printf("U	v	Magnitude	Angle\n");
	for(j=0;j<LAT_SIZE*LONG_SIZE;j++) {
		printf("%f,%f,%f,%f\n",udata[j],vdata[j],resultantMatrix[j],resultantAngle[j]);
	}

	printf("(5,5)::%f,%f",udata[5 * LAT_SIZE + 5],vdata[5 * LAT_SIZE + 5]);

	get_movementData(posdataTxt,resultantMatrix,resultantAngle,udata,vdata);
*/
//--------------------------------------------------------------------------------------------------------------------------
	/*int coords_row[TIMESTEPS];
	int coords_col[TIMESTEPS];
	int *coords_rowPtr,*coords_colPtr;

	hipMalloc((void**)&coords_rowPtr,TIMESTEPS * sizeof(int));
	hipMalloc((void**)&coords_colPtr,TIMESTEPS * sizeof(int));
	hipMalloc((void**)&resultantMatrixPtr,LONG_SIZE * LAT_SIZE * sizeof(float));
	hipMalloc((void**)&resultantAnglePtr,LONG_SIZE * LAT_SIZE * sizeof(float));

	hipMemcpy(resultantMatrixPtr,resultantMatrix,LONG_SIZE * LAT_SIZE * sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(resultantAnglePtr,resultantAngle,LONG_SIZE * LAT_SIZE * sizeof(float),hipMemcpyHostToDevice);

	dim3 gridSize2(1,0,0);
	dim3 blockSize2(1,0,0);

	bird_thread<<<gridSize2,blockSize2>>>(resultantMatrix,resultantAngle,coords_row,coords_col);
	
	hipMemcpy(coords_row,coords_rowPtr,TIMESTEPS * sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(coords_row,coords_colPtr,TIMESTEPS * sizeof(int),hipMemcpyDeviceToHost);

	hipFree(coords_rowPtr);
	hipFree(coords_colPtr);
	hipFree(resultantMatrixPtr);
	hipFree(resultantAnglePtr);

	//printf("%d,%d\n",coords_row[0],coords_col[0]);
	*/

	free(udata);
	free(vdata);
	free(dirData);

	fclose(udataTxt);
	fclose(vdataTxt);
	fclose(inpCheckU);
	fclose(inpCheckV);
	fclose(posdataTxt);
//------------------------------------------------------------------------------------------------------------------------------------
	return 0;

}


float getAngleValue(float u,float v)
{

		float angle,diffAngle;

		if((v >0)&&( u >0)) {
			angle = tanf(u/v) * (180/PI);
			diffAngle = DESIREDANGLE - angle;
		}
		else if ((v > 0)&&( u < 0)){
			angle = 360 - (tanf(u/v) * (180/PI));
			diffAngle = DESIREDANGLE + (tanf(u/v) * (180/PI));
		}
		else if ((v < 0)&&( u > 0)){
			angle = 180 - (tanf(u/v) * (180/PI));
			diffAngle = DESIREDANGLE - (tanf(u/v) * (180/PI));
		}
		else if ((v < 0)&&( u < 0)){
			angle = 180 + (tanf(u/v) * (180/PI));
			diffAngle = DESIREDANGLE + (tanf(u/v) * (180/PI)) 
		}
		else if ((v == 0)&&( u > 0)){
			angle = 90;
			diffAngle = DESIREDANGLE - angle;
		}
		else if ((v == 0)&&( u < 0)){
			angle = 270;
			diffAngle = angle - DESIREDANGLE;
		}
		else if ((v > 0)&&( u == 0)){
			angle = 0;
			diffAngle = DESIREDANGLE + angle;
		}
		else if ((v < 0)&&( u == 0)){
			angle = 180;
			diffAngle = angle - DESIREDANGLE;
		}

		if(angle < 0) angle = (360 - angle);
		if(angle > 360) angle = angle - 360;

		if(diffAngle < 0) diffAngle = 360 - angle;
		if(diffAngle > 360) diffAngle = diffAngle - 360;

		return diffAngle;
}

//void get_movementData(FILE* outTxt,float* resultantMatrix,float* resultantAngle,float* udata,float* vdata)
void get_movementData(FILE* outTxt,float* udata,float* vdata)
{

	
	int pos_row,pos_col;
	//pos_row = LONG_SIZE - STARTING_ROW;
	pos_row = STARTING_ROW;
	pos_col = STARTING_COL;

	fprintf(outTxt,"%d,%d\n",pos_row,pos_col);
	//float tempAngle = 0;
	int k;
	long i,j,l;
	j=SKIP_TIMESTEPS;
	l = 0;


	float speedOrMagnitude;
	
	long skip_size = (SKIP_TIMESTEPS) * LONG_SIZE  * LAT_SIZE - 1;
	//for(i = SKIP_TIMESTEPS - 1; i<(TIMESTEPS - 1);i++) {

	i = skip_size +pos_row * LAT_SIZE + pos_col;
	//pos_row = pos_row + (int)udata[i];
	//pos_col = pos_col + (int)vdata[i];	
	//pos_row = floorf(pos_row);
	//pos_col = floorf(pos_col);

	//fprintf(outTxt,"%d,%d\n",pos_row,pos_col);
	//printf("%f,%f,%ld\n",udata[i],vdata[i],j);
	while( i <= (TIMESTEPS-1) * LAT_SIZE * LONG_SIZE ) {
		for(k=0;k<6;k++,i++,j++,l++ ) {

			//speedOrMagnitude = hypotf(udata[SKIP_TIMESTEPS * LONG_SIZE  * LAT_SIZE + LONG_SIZE * j + pos_row * LAT_SIZE + pos_col],
					//	vdata[SKIP_TIMESTEPS * LONG_SIZE  * LAT_SIZE + LONG_SIZE * j + pos_row * LAT_SIZE + pos_col]);	
			
			pos_row = pos_row + (int)(rintf(vdata[skip_size + l * LAT_SIZE * LONG_SIZE + pos_row * LAT_SIZE + pos_col] * 3.6/50));
			pos_col = pos_col + (int)(rintf(udata[skip_size + l * LAT_SIZE * LONG_SIZE + pos_row * LAT_SIZE + pos_col] * 3.6/50));	

			
			
			//printf("%ld\n",i);
			fprintf(outTxt,"%d,%d\n",pos_row,pos_col); 
			//printf("%f,%f,%ld\n",udata[skip_size  + l * LAT_SIZE * LONG_SIZE + pos_row * LAT_SIZE + pos_col],vdata[skip_size  + l * LAT_SIZE * LONG_SIZE + pos_row * LAT_SIZE + pos_col],j);
			//i = i + 1;
			
		}
		i--;
		i = i-6;
		i = i + 24 * LAT_SIZE * LONG_SIZE;
		j--;
		j = j - 6;
		j += 24;
		l--;
		l -= 6;
		l += 24;
	}
	
}





