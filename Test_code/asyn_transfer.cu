#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define N		(1024*1024)
#define FULL_SIZE	(N*20)
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__global__ void kernel(int *a,int *b,int *c);


static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}




int main() 
{

	hipDeviceProp_t prop;
	int whichDevice;
	HANDLE_ERROR(hipGetDevice(&whichDevice));
	HANDLE_ERROR(hipGetDeviceProperties(&prop,whichDevice));


	//hipGetDevice(&whichDevice);
	//hipGetDeviceProperties(&prop,whichDevice);
	if(!prop.deviceOverlap){
		printf("Device does not handle overlaps so streams are not possible\n");
		return 0;
	}


	hipStream_t stream1;
	HANDLE_ERROR(hipStreamCreate(&stream1));	
	//hipStreamCreate(&stream1);
	int *h_a,*h_b,*h_c;
	int *d_a,*d_b,*d_c;


	HANDLE_ERROR(hipMalloc((void**)&d_a,N*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_b,N*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_c,N*sizeof(int)));
/*
	hipMalloc((void**)&d_a,N*sizeof(int));
	hipMalloc((void**)&d_b,N*sizeof(int));
	hipMalloc((void**)&d_c,N*sizeof(int));
*/	
	HANDLE_ERROR(hipHostAlloc((void**)&h_a,N*sizeof(int),hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&h_b,N*sizeof(int),hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&h_c,N*sizeof(int),hipHostMallocDefault));
/*
	hipHostAlloc((void**)&h_a,N*sizeof(int),hipHostMallocDefault);
	hipHostAlloc((void**)&h_b,N*sizeof(int),hipHostMallocDefault);
	hipHostAlloc((void**)&h_c,N*sizeof(int),hipHostMallocDefault);
*/

	for(int i = 0;i<FULL_SIZE;i++){
		h_a[i] = rand();
		h_b[i] = rand();
	}


	for(int i=0;i<FULL_SIZE;i+=N){
		//KERN_COMPLETE = 0;
		HANDLE_ERROR(hipMemcpyAsync(d_a,h_a+i,N*sizeof(int),hipMemcpyHostToDevice,stream1));
		HANDLE_ERROR(hipMemcpyAsync(d_b,h_b+i,N*sizeof(int),hipMemcpyHostToDevice,stream1));
		
		kernel<<<N/256,256,0,stream1>>>(d_a,d_b,d_c);
		
		HANDLE_ERROR(hipMemcpyAsync(h_c+i,d_c,N*sizeof(int),hipMemcpyDeviceToHost,stream1));
		
	}

	HANDLE_ERROR(hipHostFree(h_a));
	HANDLE_ERROR(hipHostFree(h_b));
	HANDLE_ERROR(hipHostFree(h_c));
	HANDLE_ERROR(hipFree(d_a));
	HANDLE_ERROR(hipFree(d_b));
	HANDLE_ERROR(hipFree(d_c));
	
	HANDLE_ERROR(hipStreamDestroy(stream1));
	return 0;
}


__global__ void kernel(int *a,int *b,int *c){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if(idx < N){
		int idx1 = (idx +1) %256;
		int idx2 = (idx +2) %256;

		float as = (a[idx]+a[idx1]+a[idx2])/3.0f;
		float bs = (b[idx]+b[idx1]+b[idx2])/3.0f;
		c[idx] = (as +bs)/2;
		//KERN_COMPLETE = 1;
	}
}


